#include "hip/hip_runtime.h"
/*
 * NormalizeLayer.cpp
 *
 *  Created on: Apr 21, 2017
 *      Author: jkim
 */

#include "NormalizeLayer.h"
#include "MathFunctions.h"
#include "NetworkConfig.h"

using namespace std;


// divid a matrix with vector
template <typename Dtype>
__global__ void DivBsx(const int nthreads, const Dtype* A,
		const Dtype* v, const int rows, const int cols, const CBLAS_TRANSPOSE trans,
		Dtype* B) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		int c = index % cols;
		int r = (index / cols) % rows;
		if (trans == CblasNoTrans) {
			B[index] = A[index] / v[c];
		} else {
			B[index] = A[index] / v[r];
		}
	}
}

template <typename Dtype>
__global__ void MulBsx(const int nthreads, const Dtype* A,
		const Dtype* v, const int rows, const int cols, const CBLAS_TRANSPOSE trans,
		Dtype* B) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		int c = index % cols;
		int r = (index / cols) % rows;
		if (trans == CblasNoTrans) {
			B[index] = A[index] * v[c];
		} else {
			B[index] = A[index] * v[r];
		}
	}
}




template <typename Dtype>
NormalizeLayer<Dtype>::NormalizeLayer(Builder* builder)
: LearnableLayer<Dtype>(builder),
  acrossSpatial(builder->_acrossSpatial),
  channelShared(builder->_channelShared),
  scaleUpdateParam(builder->_scaleUpdateParam),
  scaleFiller(builder->_scaleFiller),
  eps(builder->_eps) {

	initialize();
}

template <typename Dtype>
NormalizeLayer<Dtype>::~NormalizeLayer() {
	if (this->isReceiver) {
		Donator<Dtype>::releaseReceiver(this->donatorID);
	} else {
		Util::clearVector(this->_params);
		Util::clearVector(this->_paramsHistory);
		Util::clearVector(this->_paramsHistory2);
	}
}

template <typename Dtype>
void NormalizeLayer<Dtype>::reshape() {
	Layer<Dtype>::_adjustInputShape();

	if (!Layer<Dtype>::_isInputShapeChanged(0))
		return;

	const vector<uint32_t>& dataShape = this->_inputData[0]->getShape();
	this->buffer_.reshape({1, dataShape[1], dataShape[2], dataShape[3]});
	this->bufferChannel_.reshape({1, dataShape[1], 1, 1});
	this->bufferSpatial_.reshape({1, 1, dataShape[2], dataShape[3]});

	if (this->acrossSpatial) {
		this->norm_.reshape({dataShape[0], 1, 1, 1});
	} else {
		this->norm_.reshape({dataShape[0], 1, dataShape[2], dataShape[3]});
	}

	uint32_t channels = dataShape[1];
	uint32_t spatialDim = dataShape[2] * dataShape[3];

	this->sumChannelMultiplier_.reshape({1, channels, 1, 1});
	this->sumChannelMultiplier_.reset_host_data(false, Dtype(1.0));
	this->sumSpatialMultiplier_.reshape({1, 1, dataShape[2], dataShape[3]});
	this->sumSpatialMultiplier_.reset_host_data(false, Dtype(1.0));

	assert(this->_paramsInitialized[0] == false);
	// channel 무관하게 single scale 사용
	if (this->channelShared) {
		this->_params[0]->reshape({1, 1, 1, 1});
		this->_paramsHistory[0]->reshape({1, 1, 1, 1});
		this->_paramsHistory2[0]->reshape({1, 1, 1, 1});
	}
	// channel별 별도 scale 사용
	else {
		this->_params[0]->reshape({1, 1, 1, channels});
		this->_paramsHistory[0]->reshape({1, 1, 1, channels});
		this->_paramsHistory2[0]->reshape({1, 1, 1, channels});
	}
	this->scaleFiller.fill(this->_params[0]);
	this->_paramsInitialized[0] = true;
}


template <typename Dtype>
void NormalizeLayer<Dtype>::feedforward() {
	reshape();

	const Dtype* inputData = this->_inputData[0]->device_data();
	Dtype* outputData = this->_outputData[0]->mutable_device_data();
	Dtype* bufferData = this->buffer_.mutable_device_data();
	Dtype* normData;
	if (this->acrossSpatial) {
		// need to index it
		normData = this->norm_.mutable_host_data();
	} else {
		// add eps to avoid overflow
		this->norm_.reset_device_data(false, this->eps);
		normData = this->norm_.mutable_device_data();
	}

	const Dtype* scale;
	if (this->channelShared) {
		// vector 연산을 할 필요가 없어 host에서 scalar norm을 계산함.
		scale = this->_params[0]->host_data();
	} else {
		scale = this->_params[0]->device_data();
	}

	const Dtype* sumChannelMultiplier = this->sumChannelMultiplier_.device_data();
	int num = this->_inputData[0]->getShape(0);
	int dim = this->_inputData[0]->getCount() / num;
	int spatialDim = this->_inputData[0]->getShape(2) * this->_inputData[0]->getShape(3);
	int channels = this->_inputData[0]->getShape(1);

	for (int n = 0; n < num; n++) {
		// bufferData = inputData^2
		soooa_gpu_powx<Dtype>(dim, inputData, Dtype(2), bufferData);
		// 이미지 하나 전체에 대해 norm 적용
		if (this->acrossSpatial) {
			Dtype normsqr;
			soooa_gpu_asum<Dtype>(dim, bufferData, &normsqr);
			// add eps to avoid overflow
			normData[n] = pow(normsqr + this->eps, Dtype(0.5));
			soooa_gpu_scale<Dtype>(dim, Dtype(1.0 / normData[n]), inputData, outputData);
		}
		// 채널간 spatialDim 단위로 norm 적용
		else {
			// compute norm
			soooa_gpu_gemv<Dtype>(CblasTrans, channels, spatialDim, Dtype(1.0), bufferData,
					sumChannelMultiplier, Dtype(1.0), normData);
			soooa_gpu_powx<Dtype>(spatialDim, normData, Dtype(0.5), normData);
			// scale the layer
			DivBsx<Dtype><<<SOOOA_GET_BLOCKS(dim), SOOOA_CUDA_NUM_THREADS>>>(
					dim, inputData, normData, channels, spatialDim, CblasNoTrans, outputData);
			CUDA_POST_KERNEL_CHECK;
			normData += spatialDim;
		}

		// scale the output
		if (this->channelShared) {
			soooa_gpu_scal<Dtype>(dim, scale[0], outputData);
		} else {
			MulBsx<Dtype><<<SOOOA_GET_BLOCKS(dim), SOOOA_CUDA_NUM_THREADS>>>(
					dim, outputData, scale, channels, spatialDim, CblasTrans, outputData);
			CUDA_POST_KERNEL_CHECK;
		}
		inputData += dim;
		outputData += dim;
	}
}



template <typename Dtype>
void NormalizeLayer<Dtype>::backpropagation() {
	const Dtype* outputGrad = this->_outputData[0]->device_grad();
	const Dtype* outputData = this->_outputData[0]->device_data();
	const Dtype* inputData = this->_inputData[0]->device_data();
	Dtype* inputGrad = this->_inputData[0]->mutable_device_grad();

	const Dtype* normData;
	if (this->acrossSpatial) {
		// need to index it
		normData = this->norm_.host_data();
	} else {
		normData = this->norm_.device_data();
	}

	const Dtype* scale;
	if (this->channelShared) {
		scale = this->_params[0]->host_data();
	} else {
		scale = this->_params[0]->device_data();
	}

	Dtype* bufferData = this->buffer_.mutable_device_data();
	Dtype* bufferChannel = this->bufferChannel_.mutable_device_data();
	Dtype* bufferSpatial = this->bufferSpatial_.mutable_device_data();
	const Dtype* sumChannelMultiplier = this->sumChannelMultiplier_.device_data();
	const Dtype* sumSpatialMultiplier = this->sumSpatialMultiplier_.device_data();

	int count = this->_outputData[0]->getCount();
	int num = this->_outputData[0]->getShape(0);
	int dim = count / num;
	int spatialDim = this->_outputData[0]->getShape(2) * this->_outputData[0]->getShape(3);
	int channels = this->_outputData[0]->getShape(1);

	// propagate to param
	if (this->channelShared) {
		Dtype* scaleGrad = this->_params[0]->mutable_host_grad();
		Dtype a;
		soooa_gpu_dot<Dtype>(count, outputData, outputGrad, &a);
		scaleGrad[0] += a / scale[0];
	} else {
		Dtype* scaleGrad = this->_params[0]->mutable_device_grad();
		for (int n = 0; n < num; n++) {
			// compute a
			soooa_gpu_mul<Dtype>(dim, outputData + n * dim, outputGrad + n *dim, bufferData);
			soooa_gpu_gemv<Dtype>(CblasNoTrans, channels, spatialDim, Dtype(1.0),
					bufferData, sumSpatialMultiplier, Dtype(0.0),
					bufferChannel);
			// store a / scale[i] in bufferData temporary
			soooa_gpu_div<Dtype>(channels, bufferChannel, scale, bufferChannel);
			soooa_gpu_add<Dtype>(channels, bufferChannel, scaleGrad, scaleGrad);
		}
	}

	// propagate to bottom
	if (this->_propDown[0]) {
		for (int n = 0; n < num; n++) {
			if (this->acrossSpatial) {
				Dtype a;
				soooa_gpu_dot<Dtype>(dim, inputData, outputGrad, &a);
				soooa_gpu_scale<Dtype>(dim, a / normData[n] / normData[n], inputData,
						inputGrad);
				soooa_gpu_sub<Dtype>(dim, outputGrad, inputGrad, inputGrad);
				soooa_gpu_scale<Dtype>(dim, Dtype(1.0 / normData[n]), inputGrad, inputGrad);
			} else {
				// dot product between inputData and outputGrad
				soooa_gpu_mul<Dtype>(dim, inputData, outputGrad, bufferData);
				soooa_gpu_gemv<Dtype>(CblasTrans, channels, spatialDim, Dtype(1.0),
						bufferData, sumChannelMultiplier, Dtype(0.0),
						bufferSpatial);
				// scale bottomGrad
				MulBsx<Dtype><<<SOOOA_GET_BLOCKS(dim), SOOOA_CUDA_NUM_THREADS>>>(
						dim, inputData, bufferSpatial, channels, spatialDim,
						CblasNoTrans, inputGrad);
				CUDA_POST_KERNEL_CHECK;
				// divide by square of norm
				soooa_gpu_powx<Dtype>(spatialDim, normData, Dtype(2.0), bufferSpatial);
				DivBsx<Dtype> <<<SOOOA_GET_BLOCKS(dim), SOOOA_CUDA_NUM_THREADS>>>(
						dim, inputGrad, bufferSpatial, channels, spatialDim,
						CblasNoTrans, inputGrad);
				CUDA_POST_KERNEL_CHECK;
				soooa_gpu_sub<Dtype>(dim, outputGrad, inputGrad, inputGrad);
				// divide by norm
				DivBsx<Dtype><<<SOOOA_GET_BLOCKS(dim), SOOOA_CUDA_NUM_THREADS>>>(
						dim, inputGrad, normData, channels, spatialDim, CblasNoTrans,
						inputGrad);
				CUDA_POST_KERNEL_CHECK;
				normData += spatialDim;
			}
			// scald the grad
			if (this->channelShared) {
				soooa_gpu_scal<Dtype>(dim, scale[0], inputGrad);
			} else {
				MulBsx<Dtype><<<SOOOA_GET_BLOCKS(dim), SOOOA_CUDA_NUM_THREADS>>>(
						dim, inputGrad, scale, channels, spatialDim, CblasTrans,
						inputGrad);
				CUDA_POST_KERNEL_CHECK;
			}
			inputData += dim;
			outputGrad += dim;
			inputGrad += dim;
		}
	}
}

template <typename Dtype>
void NormalizeLayer<Dtype>::update() {
	const uint32_t weightSize = this->_params[0]->getCount();
	const Dtype regScale = this->networkConfig->_weightDecay *
			this->scaleUpdateParam.decay_mult;
	const Dtype learnScale = this->networkConfig->getLearningRate() *
			this->scaleUpdateParam.lr_mult;

	const Dtype epsilon = this->networkConfig->_epsilon;
	const Dtype decayRate = this->networkConfig->_decayRate;
	const Dtype beta1 = this->networkConfig->_beta1;
	const Dtype beta2 = this->networkConfig->_beta2;

	_updateParam(weightSize, regScale, learnScale, epsilon, decayRate, beta1, beta2,
		this->_paramsHistory[0], this->_paramsHistory2[0], this->_params[0]);
}

template <typename Dtype>
void NormalizeLayer<Dtype>::_updateParam(const uint32_t paramSize, const Dtype regScale,
    const Dtype learnScale, const Dtype epsilon, const Dtype decayRate, const Dtype beta1,
    const Dtype beta2, Data<Dtype>* dataHistory, Data<Dtype>* dataHistory2,
    Data<Dtype>* data) {

	const uint32_t batches = this->_inputShape[0][0];
	const Dtype momentum = this->networkConfig->_momentum;
	const Dtype negativeOne = -1.0;

    if (!Worker<Dtype>::isSingle())
        data->mutable_host_grad();
	Dtype* d_paramGrad = data->mutable_device_grad();
	Dtype* d_paramData = data->mutable_device_data();
	Dtype* d_paramHistoryData = dataHistory->mutable_device_data();

    // (2) apply optimizer
    Optimizer opt = this->networkConfig->_optimizer;
    assert(opt == Optimizer::Momentum);

	soooa_gpu_axpy(static_cast<int>(paramSize), regScale, d_paramData, d_paramGrad);
	soooa_gpu_axpby(static_cast<int>(paramSize), learnScale, d_paramGrad, momentum,
			d_paramHistoryData);
	soooa_copy(static_cast<int>(paramSize), d_paramHistoryData, d_paramGrad);

	// update
	soooa_gpu_axpy(static_cast<int>(paramSize), negativeOne, d_paramGrad, d_paramData);
}


template <typename Dtype>
void NormalizeLayer<Dtype>::applyChanges(LearnableLayer<Dtype> *targetLayer) {
    const uint32_t paramSize = this->_params[0]->getCount();
    NormalizeLayer<Dtype>* _targetLayer = (NormalizeLayer<Dtype>*)targetLayer;

    _targetLayer->_params[0]->add_device_grad(this->_params[0]);
}

template <typename Dtype>
void NormalizeLayer<Dtype>::syncParams(LearnableLayer<Dtype> *targetLayer) {
    const uint32_t paramSize = this->_params[0]->getCount();
    NormalizeLayer<Dtype>* _targetLayer = (NormalizeLayer<Dtype>*)targetLayer;

    this->_params[0]->set_device_grad(_targetLayer->_params[0]);
}



template <typename Dtype>
void NormalizeLayer<Dtype>::initialize() {

	this->_params.resize(1);
	this->_paramsHistory.resize(1);
	this->_paramsHistory2.resize(1);
	this->_params[0] = new Data<Dtype>(this->name + "_scale");
	this->_paramsHistory[0] = new Data<Dtype>(this->name + "_scale_history");
	this->_paramsHistory2[0] = new Data<Dtype>(this->name + "_scale_history2");

	this->_paramsInitialized.resize(1);
	this->_paramsInitialized[0] = false;

}



template class NormalizeLayer<float>;
