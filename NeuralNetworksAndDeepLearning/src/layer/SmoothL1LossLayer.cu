#include "hip/hip_runtime.h"
/*
 * SmoothL1LossLayer.cpp
 *
 *  Created on: Nov 23, 2016
 *      Author: jkim
 */

#include <vector>

#include "SmoothL1LossLayer.h"
#include "MathFunctions.h"

#define SMOOTHL1LOSSLAYER_LOG 0

using namespace std;

template <typename Dtype>
SmoothL1LossLayer<Dtype>::SmoothL1LossLayer(Builder* builder)
	: LossLayer<Dtype>(builder) {
	this->sigma2 = builder->_sigma * builder->_sigma;
	this->firstAxis = builder->_firstAxis;
	initialize();
}

template <typename Dtype>
SmoothL1LossLayer<Dtype>::~SmoothL1LossLayer() {
	delete diff;
	delete errors;
	delete ones;
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::reshape() {
	bool adjusted = Layer<Dtype>::_adjustInputShape();
	if (adjusted) {
		this->hasWeights = (this->_inputData.size() >= 3);
		if (this->hasWeights && this->_inputData.size() != 4) {
			cout << "If weights are used, must specify both inside and outside weights" << 
                endl;
			exit(-1);
		}

		this->_outputData[0]->reshape({1, 1, 1, 1});
#if SMOOTHL1LOSSLAYER_LOG
		printf("<%s> layer' output-0 has reshaped as: %dx%dx%dx%d\n",
				this->name.c_str(), 1, 1, 1, 1);
#endif
	}

	const uint32_t inputSize = this->_inputData.size();
	for (uint32_t i = 0; i < inputSize; i++) {
		if (!Layer<Dtype>::_isInputShapeChanged(i))
			continue;

		const vector<uint32_t>& inputDataShape = this->_inputData[i]->getShape();
		this->_inputShape[i] = inputDataShape;

		// rpn_bbox_pred
		if (i == 0) {
			this->diff->reshape(inputDataShape);
			this->errors->reshape(inputDataShape);
			// vector of ones used to sum
			this->ones->reshape(inputDataShape);
			this->ones->reset_host_data(false, 1.0f);

		}
		// rpn_bbox_targets
		else if (i == 1) {
			// XXX: FullyConnectedLayer의 output이 (batches, 1, rows, 1)의 현 구조를 반영,
			// 강제로 bbox_targets의 shape를 조정
			if (this->_inputData[0]->getShape() != this->_inputData[1]->getShape()) {
				this->_inputData[1]->reshape({this->_inputData[1]->getShape(2), 1,
					this->_inputData[1]->getShape(3), 1});
				assert(this->_inputData[0]->getShape() == this->_inputData[1]->getShape());
			}
			//assert(this->_inputData[0]->channels() == this->_inputData[1]->channels());
			//assert(this->_inputData[0]->height() == this->_inputData[1]->height());
			//assert(this->_inputData[0]->width() == this->_inputData[1]->width());
		}
		// rpn_bbox_inside_weights
		else if (i == 2) {
			if (this->hasWeights) {
				if (this->_inputData[0]->getShape() != this->_inputData[2]->getShape()) {
					this->_inputData[2]->reshape({this->_inputData[2]->getShape(2), 1,
						this->_inputData[2]->getShape(3), 1});
					assert(this->_inputData[0]->getShape() ==
							this->_inputData[2]->getShape());
				}
				//assert(this->_inputData[0]->channels() == this->_inputData[2]->channels());
				//assert(this->_inputData[0]->height() == this->_inputData[2]->height());
				//assert(this->_inputData[0]->width() == this->_inputData[2]->width());
			}
		}
		// rpn_bbox_outside_weights
		else if (i == 3) {
			if (this->hasWeights) {
				if (this->_inputData[0]->getShape() != this->_inputData[3]->getShape()) {
					this->_inputData[3]->reshape({this->_inputData[3]->getShape(2), 1,
						this->_inputData[3]->getShape(3), 1});
					assert(this->_inputData[0]->getShape() ==
							this->_inputData[3]->getShape());
				}
				//assert(this->_inputData[0]->channels() == this->_inputData[3]->channels());
				//assert(this->_inputData[0]->height() == this->_inputData[3]->height());
				//assert(this->_inputData[0]->width() == this->_inputData[3]->width());
			}
		}
	}
}


template <typename Dtype>
__global__ void SmoothL1Forward(const uint32_t n, const Dtype* in, Dtype* out,
    Dtype sigma2) {
  // f(x) = 0.5 * (sigma2 * x)^2          if |x| < 1 / sigma2 / sigma2
  //        |x| - 0.5 / sigma2 / sigma2    otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1.0 / sigma2) {
      out[index] = 0.5 * val * val * sigma2;
    } else {
      out[index] = abs_val - 0.5 / sigma2;
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::feedforward() {
	reshape();

	const uint32_t count = this->_inputData[0]->getCount();
	// prediction (inputData[0]) - target (inputData[1]) => diff
	soooa_gpu_sub(
			count,
			this->_inputData[0]->device_data(),
			this->_inputData[1]->device_data(),
			this->diff->mutable_device_data());		// d := b0 - b1

#if SMOOTHL1LOSSLAYER_LOG
	this->_printOn();
	this->_inputData[0]->print_data({}, false, -1);
	this->_inputData[1]->print_data({}, false, -1);
	this->diff->print_data({}, false, -1);
	this->_printOff();
#endif

	if (hasWeights) {

#if SMOOTHL1LOSSLAYER_LOG
		this->_printOn();
		this->_inputData[2]->print_data({}, false, -1);
		this->diff->print_data({}, false, -1);
		this->_printOff();
#endif
		// apply "inside" weights
		soooa_gpu_mul(
				count,
				this->_inputData[2]->device_data(),
				diff->device_data(),
				diff->mutable_device_data());	// d := w_in * (b0 - b1)

#if SMOOTHL1LOSSLAYER_LOG
		this->_printOn();
		this->diff->print_data({}, false, -1);
		this->_printOff();
#endif

	}

	// smoothL1Forward
	SmoothL1Forward<Dtype><<<SOOOA_GET_BLOCKS(count), SOOOA_CUDA_NUM_THREADS>>>(
	      count, diff->device_data(), errors->mutable_device_data(), this->sigma2);
	CUDA_POST_KERNEL_CHECK;

#if SMOOTHL1LOSSLAYER_LOG
	this->_printOn();
	this->diff->print_data({}, false, -1);
	this->errors->print_data({}, false, -1);
	this->_printOff();
#endif

	if (hasWeights) {

#if SMOOTHL1LOSSLAYER_LOG
		this->_printOn();
		this->_inputData[3]->print_data({}, false, -1);
		this->errors->print_data({}, false, -1);
		this->_printOff();
#endif

		// apply "outside" weights
		soooa_gpu_mul(
				count,
				this->_inputData[3]->device_data(),
				errors->device_data(),
				errors->mutable_device_data());	// d := w_out * SmoothL1(w_in * (b0 - b1))

#if SMOOTHL1LOSSLAYER_LOG
		this->_printOn();
		this->errors->print_data({}, false, -1);
		this->_printOff();
#endif
	}

	Dtype loss;
	soooa_gpu_dot(count, ones->device_data(), errors->device_data(), &loss);
	this->_outputData[0]->mutable_host_data()[0] = loss * Dtype(this->lossWeight) /
			this->_inputData[0]->getShape(this->firstAxis);
	//this->_outputData[0]->mutable_host_data()[0] = loss * Dtype(this->lossWeight);
	//cout << "smoothl1loss: " << this->_outputData[0]->host_data()[0] << endl;

#if SMOOTHL1LOSSLAYER_LOG
	this->_printOn();
	this->_outputData[0]->print_data({}, false);
	this->_printOff();
#endif
}


template <typename Dtype>
__global__ void SmoothL1Backward(const uint32_t n, const Dtype* in, Dtype* out,
    Dtype sigma2) {
  // f'(x) = sigma2 * sigma2 * x         if |x| < 1 / sigma2 / sigma2
  //       = sign(x)                   otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1.0 / sigma2) {
      out[index] = sigma2 * val;
    } else {
      out[index] = (Dtype(0) < val) - (val < Dtype(0));
    }
  }
}


template <typename Dtype>
void SmoothL1LossLayer<Dtype>::backpropagation() {
	// after forwards, diff holds w_in * (b0 - b1)
	const uint32_t count = diff->getCount();
	SmoothL1Backward<Dtype><<<SOOOA_GET_BLOCKS(count), SOOOA_CUDA_NUM_THREADS>>>(
			count, diff->device_data(), diff->mutable_device_data(), this->sigma2);
	CUDA_POST_KERNEL_CHECK;

	for (uint32_t i = 0; i < 2; i++) {
		if (this->_propDown[i]) {
			const Dtype sign = (i == 0) ? 1 : -1;
			// XXX: caffe, top[0]->cpu_diff()[0]에 대해서 set하는 부분을 찾을 수 없고
			// 현재 특수한 값이 들어 있는 것이 아닌 1의 값이 들어있어 상수 1.0f으로 대체
			//const Dtype alpha = sign * this->_outputData[0]->host_grad()[0] /
			//		this->_inputData[i]->batches();
			const Dtype alpha = sign * Dtype(1) /
					this->_inputData[i]->getShape(this->firstAxis);
			soooa_gpu_axpby(
					count,
					alpha,
					diff->device_data(),
					Dtype(0),
					this->_inputData[i]->mutable_device_grad());

			if (hasWeights) {
				// Scale by "inside" weight
				soooa_gpu_mul(
						count,
						this->_inputData[2]->device_data(),
						this->_inputData[i]->device_grad(),
						this->_inputData[i]->mutable_device_grad());

				// Scale by "outside" weight
				soooa_gpu_mul(
						count,
						this->_inputData[3]->device_data(),
						this->_inputData[i]->device_grad(),
						this->_inputData[i]->mutable_device_grad());
			}
		}
	}
}

template <typename Dtype>
Dtype SmoothL1LossLayer<Dtype>::cost() {
	return this->_outputData[0]->host_data()[0];
}




template <typename Dtype>
void SmoothL1LossLayer<Dtype>::initialize() {
	diff = new Data<Dtype>("diff");
	errors = new Data<Dtype>("errors");
	ones = new Data<Dtype>("ones");
}

template class SmoothL1LossLayer<float>;
