
#include <hip/hip_runtime.h>
/*
 * PoolingLayer.cpp
 *
 *  Created on: 2016. 5. 23.
 *      Author: jhkim
 */


#ifdef GPU_MODE

#include "PoolingLayer.h"

using namespace std;

template <typename Dtype>
void PoolingLayer<Dtype>::shape() {
	Layer<Dtype>::_adjustInputShape();

	if (!Layer<Dtype>::_isInputShapeChanged(0))
		return;

	const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
	uint32_t batches 	= inputShape[0];
	uint32_t channels 	= inputShape[1];
	uint32_t rows 		= inputShape[2];
	uint32_t cols 		= inputShape[3];

	checkCUDNN(cudnnSetTensor4dDescriptor(
			this->inputTensorDesc,
			CUDNN_TENSOR_NCHW,
			CUDNN_DATA_FLOAT,
			batches, channels, rows, cols));

	int n = 0, c = 0, h = 0, w = 0;
	checkCUDNN(cudnnGetPooling2dForwardOutputDim(
			pooling_fn->getPoolDesc(),
			this->inputTensorDesc,
			&n, &c, &h, &w));

	checkCUDNN(cudnnSetTensor4dDescriptor(
			this->outputTensorDesc,
			CUDNN_TENSOR_NCHW,
			CUDNN_DATA_FLOAT,
			n, c, h, w));

	uint32_t obatches = static_cast<uint32_t>(n);
	uint32_t ochannels = static_cast<uint32_t>(c);
	uint32_t orows = static_cast<uint32_t>(h);
	uint32_t ocols = static_cast<uint32_t>(w);

	printf("<%s> layer' output-0 has reshaped as: %dx%dx%dx%d\n",
			this->name.c_str(), obatches, ochannels, orows, ocols);

	this->_inputShape[0] = inputShape;
	this->_outputData[0]->shape({obatches, ochannels, orows, ocols});

	/*
	this->setInDimension(this->_inputData[0]->getShape());

	cudnnTensorDescriptor_t tempInputTensorDesc;
	checkCUDNN(cudnnCreateTensorDescriptor(&tempInputTensorDesc));
	checkCUDNN(cudnnSetTensor4dDescriptor(tempInputTensorDesc,
				CUDNN_TENSOR_NCHW,
				CUDNN_DATA_FLOAT,
				this->in_dim.batches, this->in_dim.channels, this->in_dim.rows, this->in_dim.cols));

	int n, c, h, w;
	checkCUDNN(cudnnGetPooling2dForwardOutputDim(pooling_fn->getPoolDesc(),
			tempInputTensorDesc,
			&n, &c, &h, &w));

	this->out_dim.batches = n;
	this->out_dim.channels = c;
	this->out_dim.rows = h;
	this->out_dim.cols = w;

	checkCUDNN(cudnnDestroyTensorDescriptor(tempInputTensorDesc));

	if(recursive) {
		HiddenLayer<Dtype>::_shape();
	}
	*/
}

template <typename Dtype>
void PoolingLayer<Dtype>::_feedforward() {
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

	this->_inputData[0]->print_data();

	pooling_fn->forward(this->inputTensorDesc, d_inputData,
			this->outputTensorDesc, d_outputData);

	this->_outputData[0]->print_data();
}

template <typename Dtype>
void PoolingLayer<Dtype>::_backpropagation() {
	this->_outputData[0]->print_data();
	this->_inputData[0]->print_data();
	/*
	if(this->_output->is_nan_grad()) {
		cout << this->name << " output gradient nan ... " << endl;
		exit(1);
	}
	*/
	const Dtype* d_outputData = this->_outputData[0]->device_data();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	Dtype* d_inputGrad = this->_inputData[0]->mutable_device_grad();
	pooling_fn->backward(this->outputTensorDesc, d_outputData, d_outputGrad,
			this->inputTensorDesc, d_inputData, d_inputGrad);

	this->_inputData[0]->print_grad();
}


template void PoolingLayer<float>::shape();
template void PoolingLayer<float>::_feedforward();
template void PoolingLayer<float>::_backpropagation();

#endif




