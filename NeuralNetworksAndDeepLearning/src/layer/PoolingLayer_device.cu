
#include <hip/hip_runtime.h>
/*
 * PoolingLayer.cpp
 *
 *  Created on: 2016. 5. 23.
 *      Author: jhkim
 */


#ifdef GPU_MODE

#include "PoolingLayer.h"

using namespace std;

template <typename Dtype>
void PoolingLayer<Dtype>::_shape(bool recursive) {
	this->setInDimension(this->_inputData[0]->getShape());

	cudnnTensorDescriptor_t tempInputTensorDesc;
	checkCUDNN(cudnnCreateTensorDescriptor(&tempInputTensorDesc));
	checkCUDNN(cudnnSetTensor4dDescriptor(tempInputTensorDesc,
				CUDNN_TENSOR_NCHW,
				CUDNN_DATA_FLOAT,
				this->in_dim.batches, this->in_dim.channels, this->in_dim.rows, this->in_dim.cols));

	int n, c, h, w;
	checkCUDNN(cudnnGetPooling2dForwardOutputDim(pooling_fn->getPoolDesc(),
			tempInputTensorDesc,
			&n, &c, &h, &w));

	this->out_dim.batches = n;
	this->out_dim.channels = c;
	this->out_dim.rows = h;
	this->out_dim.cols = w;

	checkCUDNN(cudnnDestroyTensorDescriptor(tempInputTensorDesc));

	if(recursive) {
		HiddenLayer<Dtype>::_shape();
	}
}

template <typename Dtype>
void PoolingLayer<Dtype>::_feedforward() {
	this->_inputData[0]->print_data("inputData:");
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	Dtype* d_outputData = this->_outputData[0]->mutable_device_data();
	pooling_fn->forward(this->inputTensorDesc, d_inputData,
			this->outputTensorDesc, d_outputData);
	this->_outputData[0]->print_data(this->name+string("/outputData:"));
}

template <typename Dtype>
void PoolingLayer<Dtype>::_backpropagation() {
	this->_outputData[0]->print_data("outputData:");
	this->_inputData[0]->print_data("inputData:");
	/*
	if(this->_output->is_nan_grad()) {
		cout << this->name << " output gradient nan ... " << endl;
		exit(1);
	}
	*/
	const Dtype* d_outputData = this->_outputData[0]->device_data();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	Dtype* d_inputGrad = this->_inputData[0]->mutable_device_grad();
	pooling_fn->backward(this->outputTensorDesc, d_outputData, d_outputGrad,
			this->inputTensorDesc, d_inputData, d_inputGrad);

	this->_inputData[0]->print_grad("inputGrad:");
}


template void PoolingLayer<float>::_shape(bool recursive);
template void PoolingLayer<float>::_feedforward();
template void PoolingLayer<float>::_backpropagation();

#endif




