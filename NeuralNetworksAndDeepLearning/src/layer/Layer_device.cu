
#include <hip/hip_runtime.h>
/*
 * Layer.cu
 *
 *  Created on: 2016. 8. 25.
 *      Author: jhkim
 */

#ifdef GPU_MODE

#include "Layer.h"
#include "Cuda.h"

using namespace std;

/*
template <typename Dtype>
void Layer<Dtype>::_shape(bool recursive) {

	const uint32_t inputSize = _inputData.size();
	// 입력 shape가 입력 데이터만큼 할당되지 않은 경우 해당 사이즈만큼 재할당
	if (_inputShape.size() != inputSize) {
		_inputShape.resize(inputSize);
		for (uint32_t i = 0; i < inputSize; i++) {
			_inputShape[i].resize(4);
		}
	}

	// 모든 입력 데이터에 대해
	for (uint32_t i = 0; i < inputSize; i++) {
		// 이미 shape가 동일한 경우 reshape하지 않는다.
		if (_inputData[i].shape() == _inputShape[i])
			continue;

		_outputData[i]->
	}

	printf("%15s_shape():in-%dx%dx%dx%d, out-%dx%dx%dx%d\n",
			name.c_str(), in_dim.rows, in_dim.cols, in_dim.channels, in_dim.batches,
			out_dim.rows, out_dim.cols, out_dim.channels, out_dim.batches);

	// 다른 레이어에 의해 이미 shape 처리되지 않은 Data에 대해서만 shape를 수행한다.
	if (_inputData.size() > 0 && _inputData[0]->getCount() == 0)
		_inputData[0]->shape({in_dim.batches, in_dim.channels, in_dim.rows, in_dim.cols});
	if (_outputData.size() > 0 &&_outputData[0]->getCount() == 0)
		_outputData[0]->shape({out_dim.batches, out_dim.channels, out_dim.rows, out_dim.cols});



	checkCUDNN(cudnnSetTensor4dDescriptor(inputTensorDesc,
			CUDNN_TENSOR_NCHW,
			CUDNN_DATA_FLOAT,
			in_dim.batches, in_dim.channels, in_dim.rows, in_dim.cols));
	checkCUDNN(cudnnSetTensor4dDescriptor(outputTensorDesc,
			CUDNN_TENSOR_NCHW,
			CUDNN_DATA_FLOAT,
			out_dim.batches, out_dim.channels, out_dim.rows, out_dim.cols));
}
*/

template <typename Dtype>
void Layer<Dtype>::_clearShape() {
	//checkCUDNN(cudnnDestroyTensorDescriptor(inputTensorDesc));
	//checkCUDNN(cudnnDestroyTensorDescriptor(outputTensorDesc));

	//delete _input;
	//delete _output;
	//_input = NULL;
	//_output = NULL;
	//inputTensorDesc = NULL;
	//outputTensorDesc = NULL;
}

//template void Layer<float>::_shape(bool recursive);
template void Layer<float>::_clearShape();

#endif



































