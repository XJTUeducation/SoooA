
#include <hip/hip_runtime.h>
/*
 * ConvLayer.cpp
 *
 *  Created on: 2016. 5. 23.
 *      Author: jhkim
 */


#ifdef GPU_MODE
#include "ConvLayer.h"
#include "FullyConnectedLayer.h"
#include "Util.h"
#include "Exception.h"
#include "NetworkConfig.h"
#include "cuda_runtime.h"
#include <algorithm>

using namespace std;

/**
 * dst array에 src array를 더한다.
 *
 * @param dst dst array, dst + src가 저장이 될 장소
 * @param src src array
 * @param N The number of elements in the array.
 */
template <typename Dtype>
__global__ void AddArrayOfConvLayer(Dtype* dst, const Dtype* src, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;

	dst[idx] = dst[idx] + src[idx];
}

template <typename Dtype>
ConvLayer<Dtype>::~ConvLayer() {
	delete _params[ParamType::Filter];
	delete _params[ParamType::Bias];
	_params.clear();

	delete _paramsHistory[ParamType::Filter];
	delete _paramsHistory[ParamType::Bias];
	_paramsHistory.clear();

	delete _preActivation;

	if(d_workspace) checkCudaErrors(cudaFree(d_workspace));

	checkCUDNN(cudnnDestroyTensorDescriptor(inputTensorDesc));
	checkCUDNN(cudnnDestroyTensorDescriptor(outputTensorDesc));
	checkCUDNN(cudnnDestroyTensorDescriptor(biasTensorDesc));
	checkCUDNN(cudnnDestroyFilterDescriptor(filterDesc));
	checkCUDNN(cudnnDestroyConvolutionDescriptor(convDesc));

	ActivationFactory<Dtype>::destory(activation_fn);
}

template <typename Dtype>
void ConvLayer<Dtype>::initialize(filter_dim filter_d, update_param weight_update_param, update_param bias_update_param,
		param_filler<Dtype> weight_filler, param_filler<Dtype> bias_filler, typename Activation<Dtype>::Type activationType) {

	this->type = Layer<Dtype>::Conv;
	this->filter_d = filter_d;

	this->weight_update_param = weight_update_param;
	this->bias_update_param = bias_update_param;
	this->weight_filler = weight_filler;
	this->bias_filler = bias_filler;

	const int filter_size = filter_d.size();

	this->_params.resize(2);
	this->_params[Filter] = new Data<Dtype>("Filter");
	this->_params[Bias] = new Data<Dtype>("Bias");
	this->_params[Filter]->shape({filter_d.filters, filter_d.channels, filter_d.rows, filter_d.cols});
	this->_params[Bias]->shape({filter_d.filters, 1, 1, 1});

	this->_paramsHistory.resize(2);
	this->_paramsHistory[Filter] = new Data<Dtype>("FilterHistory");
	this->_paramsHistory[Bias] = new Data<Dtype>("BiasHistory");
	this->_paramsHistory[Filter]->shape({filter_d.filters, filter_d.channels, filter_d.rows, filter_d.cols});
	this->_paramsHistory[Bias]->shape({filter_d.filters, 1, 1, 1});

	this->_preActivation = new Data<Dtype>("PreActivation");

	checkCUDNN(cudnnCreateTensorDescriptor(&inputTensorDesc));
	checkCUDNN(cudnnCreateTensorDescriptor(&outputTensorDesc));
	checkCUDNN(cudnnCreateTensorDescriptor(&biasTensorDesc));
	checkCUDNN(cudnnCreateFilterDescriptor(&filterDesc));
	checkCUDNN(cudnnCreateConvolutionDescriptor(&convDesc));

	checkCUDNN(cudnnSetTensor4dDescriptor(biasTensorDesc,
			CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT,
			1, filter_d.filters, 1, 1));

	checkCUDNN(cudnnSetFilter4dDescriptor(filterDesc,
			CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW,
			filter_d.filters, filter_d.channels, filter_d.rows, filter_d.cols));

	//int pad = (filter_d.rows-1)/2;
	checkCUDNN(cudnnSetConvolution2dDescriptor(convDesc,
			filter_d.pad, filter_d.pad, filter_d.stride, filter_d.stride, 1, 1,
			CUDNN_CROSS_CORRELATION));

	this->activation_fn = ActivationFactory<Dtype>::create(activationType);
}

template <typename Dtype>
void ConvLayer<Dtype>::reshape() {
	Layer<Dtype>::_adjustInputShape();

	if (!Layer<Dtype>::_isInputShapeChanged(0))
		return;

	const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
	uint32_t batches 	= inputShape[0];
	uint32_t channels 	= inputShape[1];
	uint32_t rows 		= inputShape[2];
	uint32_t cols 		= inputShape[3];

	checkCUDNN(cudnnSetTensor4dDescriptor(
			this->inputTensorDesc,
			CUDNN_TENSOR_NCHW,
			CUDNN_DATA_FLOAT,
			batches, channels, rows, cols));

	int n = 0, c = 0, h = 0, w = 0;
	checkCUDNN(cudnnGetConvolution2dForwardOutputDim(
			convDesc,
			this->inputTensorDesc,
			filterDesc,
			&n, &c, &h, &w));

	checkCUDNN(cudnnSetTensor4dDescriptor(
			this->outputTensorDesc,
			CUDNN_TENSOR_NCHW,
			CUDNN_DATA_FLOAT,
			n, c, h, w));

	const uint32_t obatches = static_cast<uint32_t>(n);
	const uint32_t ochannels = static_cast<uint32_t>(c);
	const uint32_t orows = static_cast<uint32_t>(h);
	const uint32_t ocols = static_cast<uint32_t>(w);

	printf("<%s> layer' output-0 has reshaped as: %dx%dx%dx%d\n",
			this->name.c_str(), obatches, ochannels, orows, ocols);

	this->_inputShape[0] = inputShape;
	this->_preActivation->shape({obatches, ochannels, orows, ocols});
	this->_outputData[0]->shape({obatches, ochannels, orows, ocols});


	//int u_in = this->in_dim.unitsize();
	//int u_out = this->out_dim.unitsize();
	//int b_in = this->in_dim.batchsize();
	//int b_out = this->out_dim.batchsize();
	int u_in = channels * rows * cols;
	int u_out = c * h * w;
	int b_in = batches * channels * rows * cols;
	int b_out = n * c * h * w;

	weight_filler.fill(_params[Filter]);
	bias_filler.fill(_params[Bias]);


	size_t convFwdWorkspaceSize;
	size_t convBwdFilterWorkspaceSize;
	size_t convBwdDataWorkspaceSize;

	// forward algorithm
	checkCUDNN(cudnnGetConvolutionForwardAlgorithm(
			Cuda::cudnnHandle,
			this->inputTensorDesc,
			filterDesc,
			convDesc,
			this->outputTensorDesc,
			CUDNN_CONVOLUTION_FWD_PREFER_FASTEST,
			8<<20,
			&convFwdAlgo));

	checkCUDNN(cudnnGetConvolutionForwardWorkspaceSize(
			Cuda::cudnnHandle,
			this->inputTensorDesc,
			filterDesc,
			convDesc,
			this->outputTensorDesc,
			convFwdAlgo,
			&convFwdWorkspaceSize));

	// backward filter algorithm
	checkCUDNN(cudnnGetConvolutionBackwardFilterAlgorithm(
			Cuda::cudnnHandle,
			this->inputTensorDesc,
			this->outputTensorDesc,
			convDesc,
			filterDesc,
			CUDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
			8<<20,
			&convBwdFilterAlgo));

	checkCUDNN(cudnnGetConvolutionBackwardFilterWorkspaceSize(
			Cuda::cudnnHandle,
			this->inputTensorDesc,
			this->outputTensorDesc,
			convDesc,
			filterDesc,
			convBwdFilterAlgo,
			&convBwdFilterWorkspaceSize));

	// backward data algorithm
	checkCUDNN(cudnnGetConvolutionBackwardDataAlgorithm(
			Cuda::cudnnHandle,
			filterDesc,
			this->outputTensorDesc,
			convDesc,
			this->inputTensorDesc,
			CUDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
			8<<20,
			&convBwdDataAlgo));

	checkCUDNN(cudnnGetConvolutionBackwardDataWorkspaceSize(
			Cuda::cudnnHandle,
			filterDesc,
			this->outputTensorDesc,
			convDesc,
			this->inputTensorDesc,
			convBwdDataAlgo,
			&convBwdDataWorkspaceSize));

	workspaceSize = 0;
	workspaceSize = max(workspaceSize, convFwdWorkspaceSize);
	workspaceSize = max(workspaceSize, convBwdFilterWorkspaceSize);
	workspaceSize = max(workspaceSize, convBwdDataWorkspaceSize);

	d_workspace = 0;
	if(workspaceSize > 0) {
		checkCudaErrors(Util::ucudaMalloc(&d_workspace, workspaceSize));
	}

	/*
	this->setInDimension(this->_inputData[0]->getShape());

	cudnnTensorDescriptor_t tempInputTensorDesc;
	checkCUDNN(cudnnCreateTensorDescriptor(&tempInputTensorDesc));
	checkCUDNN(cudnnSetTensor4dDescriptor(tempInputTensorDesc,
				CUDNN_TENSOR_NCHW,
				CUDNN_DATA_FLOAT,
				this->in_dim.batches, this->in_dim.channels, this->in_dim.rows, this->in_dim.cols));

	int n = 0, c = 0, h = 0, w = 0;
	checkCUDNN(cudnnGetConvolution2dForwardOutputDim(convDesc,
			tempInputTensorDesc, filterDesc,
			&n, &c, &h, &w));

	this->out_dim.batches = n;
	this->out_dim.channels = c;
	this->out_dim.rows = h;
	this->out_dim.cols = w;

	checkCUDNN(cudnnDestroyTensorDescriptor(tempInputTensorDesc));

	if(recursive) {
		HiddenLayer<Dtype>::_shape();
	}

	int u_in = this->in_dim.unitsize();
	int u_out = this->out_dim.unitsize();
	int b_in = this->in_dim.batchsize();
	int b_out = this->out_dim.batchsize();

	weight_filler.fill(_params[Filter]);
	bias_filler.fill(_params[Bias]);


	_params[Filter]->print_data(this->name + " filter: ");



	_preActivation->shape({this->out_dim.batches, this->out_dim.channels, this->out_dim.rows, this->out_dim.cols});

	size_t convFwdWorkspaceSize;
	size_t convBwdFilterWorkspaceSize;
	size_t convBwdDataWorkspaceSize;
	// forward algorithm
	checkCUDNN(cudnnGetConvolutionForwardAlgorithm(Cuda::cudnnHandle,
			this->inputTensorDesc, filterDesc, convDesc, this->outputTensorDesc,
			CUDNN_CONVOLUTION_FWD_PREFER_FASTEST, 8<<20, &convFwdAlgo));
			//CUDNN_CONVOLUTION_FWD_NO_WORKSPACE, 0, &convFwdAlgo));
	checkCUDNN(cudnnGetConvolutionForwardWorkspaceSize(Cuda::cudnnHandle,
			this->inputTensorDesc, filterDesc, convDesc, this->outputTensorDesc,
			convFwdAlgo, &convFwdWorkspaceSize));


	// backward filter algorithm
	checkCUDNN(cudnnGetConvolutionBackwardFilterAlgorithm(Cuda::cudnnHandle,
			this->inputTensorDesc, this->outputTensorDesc, convDesc, filterDesc,
			CUDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 8<<20, &convBwdFilterAlgo));
			//CUDNN_CONVOLUTION_BWD_FILTER_NO_WORKSPACE, 32<<20, &convBwdFilterAlgo));
	checkCUDNN(cudnnGetConvolutionBackwardFilterWorkspaceSize(Cuda::cudnnHandle,
			this->inputTensorDesc, this->outputTensorDesc, convDesc, filterDesc,
			convBwdFilterAlgo, &convBwdFilterWorkspaceSize));


	// backward data algorithm
	checkCUDNN(cudnnGetConvolutionBackwardDataAlgorithm(Cuda::cudnnHandle,
			filterDesc, this->outputTensorDesc, convDesc, this->inputTensorDesc,
			CUDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 8<<20, &convBwdDataAlgo));
			//CUDNN_CONVOLUTION_BWD_DATA_NO_WORKSPACE, 32<<20, &convBwdDataAlgo));
	checkCUDNN(cudnnGetConvolutionBackwardDataWorkspaceSize(Cuda::cudnnHandle,
			filterDesc, this->outputTensorDesc, convDesc, this->inputTensorDesc,
			convBwdDataAlgo, &convBwdDataWorkspaceSize));

	workspaceSize = 0;
	workspaceSize = max(workspaceSize, convFwdWorkspaceSize);
	workspaceSize = max(workspaceSize, convBwdFilterWorkspaceSize);
	workspaceSize = max(workspaceSize, convBwdDataWorkspaceSize);
	//cout << workspaceSize << ", " << convFwdWorkspaceSize << ", " << convBwdFilterWorkspaceSize << ", " << convBwdDataWorkspaceSize << endl;

	d_workspace = 0;
	if(workspaceSize > 0) {
		//cout << "workspaceSize: " << workspaceSize << endl;
		checkCudaErrors(Util::ucudaMalloc(&d_workspace, workspaceSize));
	}
	*/
}

template <typename Dtype>
void ConvLayer<Dtype>::_clearShape() {
	delete _params[0];
	delete _params[1];
	//_params.clear();

	delete _paramsHistory[0];
	delete _paramsHistory[1];
	//_paramsHistory.clear();

	delete _preActivation;

	if(d_workspace) {
		checkCudaErrors(cudaFree(d_workspace));
		d_workspace = 0;
	}

	HiddenLayer<Dtype>::_clearShape();
}


template <typename Dtype>
void ConvLayer<Dtype>::update() {
	// update filters ...
	const uint32_t weightSize = filter_d.size();
	const Dtype regScale = this->networkConfig->_weightDecay * weight_update_param.decay_mult;
	const Dtype learnScale = this->networkConfig->getLearningRate() * weight_update_param.lr_mult;
	_updateParam(weightSize, regScale, learnScale, _paramsHistory[Filter], _params[Filter]);

	// update biases ...
	const uint32_t biasSize = filter_d.filters;
	const Dtype regScale_b = this->networkConfig->_weightDecay * bias_update_param.decay_mult;
	const Dtype learnScale_b = this->networkConfig->getLearningRate() * bias_update_param.lr_mult;
	_updateParam(biasSize, regScale_b, learnScale_b, _paramsHistory[Bias], _params[Bias]);
}




template <typename Dtype>
void ConvLayer<Dtype>::_updateParam(const uint32_t paramSize, const Dtype regScale, const Dtype learnScale, Data<Dtype>* dataHistory, Data<Dtype>* data) {
	const uint32_t batches = this->_inputData[0]->getShape(0);
	const Dtype normScale = 1.0/batches;
	const Dtype momentum = this->networkConfig->_momentum;
	const Dtype negativeOne = -1.0;

    data->mutable_host_grad();
	Dtype* d_paramGrad = data->mutable_device_grad();   // should update grad
	Dtype* d_paramData = data->mutable_device_data();
	Dtype* d_paramHistoryData = dataHistory->mutable_device_data();

	checkCudaErrors(cublasSscal(Cuda::cublasHandle, static_cast<int>(paramSize), &normScale, d_paramGrad, 1));							// normalized by batch size
	checkCudaErrors(cublasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize), &regScale, d_paramData, 1, d_paramGrad, 1));			// regularize
	checkCudaErrors(cublasSscal(Cuda::cublasHandle, static_cast<int>(paramSize), &momentum, d_paramHistoryData, 1));					//
	checkCudaErrors(cublasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize), &learnScale, d_paramGrad, 1, d_paramHistoryData, 1));	// momentum
	checkCudaErrors(cublasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize), &negativeOne, d_paramHistoryData, 1, d_paramData, 1));	// update

}

template <typename Dtype>
void ConvLayer<Dtype>::applyChanges(LearnableLayer<Dtype> *targetLayer) {
	const uint32_t weightSize = filter_d.size();
	const uint32_t biasSize = filter_d.filters;
    ConvLayer<Dtype>* _targetLayer = (ConvLayer<Dtype>*)targetLayer;

    int blockSize = BW;
    int gridSize = (weightSize + blockSize -1) / blockSize;

    AddArrayOfConvLayer<<<gridSize, blockSize>>>(
        _targetLayer->_params[Filter]->mutable_device_grad(),
        _params[Filter]->device_grad(), weightSize);

    gridSize = (biasSize + blockSize -1) / blockSize;

    AddArrayOfConvLayer<<<gridSize, blockSize>>>(
        _targetLayer->_params[Bias]->mutable_device_grad(),
        _params[Bias]->device_grad(), biasSize);
}

template <typename Dtype>
void ConvLayer<Dtype>::syncParams(LearnableLayer<Dtype> *targetLayer) {
	const uint32_t weightSize = filter_d.size();
	const uint32_t biasSize = filter_d.filters;
    ConvLayer<Dtype>* _targetLayer = (ConvLayer<Dtype>*)targetLayer;

    memcpy(_params[Filter]->mutable_host_grad(), _targetLayer->_params[Filter]->host_grad(),
        weightSize);
    memcpy(_params[Bias]->mutable_host_grad(), _targetLayer->_params[Bias]->host_grad(),
        biasSize);
#if 0
    for (uint32_t paramIdx = 0; paramIdx < weightSize; paramIdx++) {
        _params[Filter]->mutable_host_grad()[paramIdx] = 
            _targetLayer->_params[Filter]->host_grad()[paramIdx];
    }

    for (uint32_t paramIdx = 0; paramIdx < biasSize; paramIdx++) {
        _params[Bias]->mutable_host_grad()[paramIdx] = 
            _targetLayer->_params[Bias]->host_grad()[paramIdx];
    }
#endif
}

template <typename Dtype>
void ConvLayer<Dtype>::syncMutableMem() {
	_params[Filter]->mutable_device_grad();
	_params[Filter]->host_grad();
	_params[Bias]->mutable_device_grad();
	_params[Bias]->host_data();
}













template <typename Dtype>
void ConvLayer<Dtype>::feedforward() {
	reshape();

	_computeFiltersConvolutionData();
	_computeActivationData();
}



template <typename Dtype>
void ConvLayer<Dtype>::_computeFiltersConvolutionData() {
	// Apply filters to input data
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	const Dtype* d_filtersData = _params[Filter]->device_data();
	Dtype* d_preActivationData = _preActivation->mutable_device_data();

	this->_inputData[0]->print_data();
	_params[Filter]->print_data();

	checkCUDNN(cudnnConvolutionForward(Cuda::cudnnHandle,
			&Cuda::alpha, this->inputTensorDesc, d_inputData, filterDesc, d_filtersData, convDesc, convFwdAlgo, d_workspace, workspaceSize,
			&Cuda::beta, this->outputTensorDesc, d_preActivationData));

	_preActivation->print_data();

	// Add bias to filtered input data
	_params[Bias]->print_data();

	const Dtype* d_biasesData = _params[Bias]->device_data();

	checkCUDNN(cudnnAddTensor(Cuda::cudnnHandle,
			&Cuda::alpha, biasTensorDesc, d_biasesData,
			&Cuda::alpha, this->outputTensorDesc, d_preActivationData));

	_preActivation->print_data();
}

template <typename Dtype>
void ConvLayer<Dtype>::_computeActivationData() {
	// Activate filtered result
	const Dtype* d_preActivationData = _preActivation->device_data();
	Dtype* d_output = this->_outputData[0]->mutable_device_data();

	_preActivation->print_data();

	if (activation_fn)
		activation_fn->forward(this->outputTensorDesc, d_preActivationData, d_output);

	this->_outputData[0]->print_data();
}



template <typename Dtype>
void ConvLayer<Dtype>::backpropagation() {
	// 여러 source로부터 delta값이 모두 모이면 dw, dx 계산

	/*
	//Util::printDeviceData(d_delta_output, out_dim.rows, out_dim.cols, out_dim.channels, out_dim.batches, "delta_output:");
	//Util::printDeviceData(d_output, out_dim.rows, out_dim.cols, out_dim.channels, out_dim.batches, "output:");
	this->_output->print_grad("delta_output:");
	this->_output->print_data("output:");

	const Dtype* d_output = this->_output->device_data();
	const Dtype* d_delta_output = this->_output->device_grad();
	const Dtype* d_z = _preActivation->device_data();
	Dtype* d_delta = _preActivation->mutable_device_grad();

	//activation_fn->backward(d_output, d_delta_output, d_z, d_delta, outputTensorDesc);
	activation_fn->backward(this->outputTensorDesc, d_output, d_delta_output, d_z, d_delta);
	//Util::printDeviceData(d_delta, out_dim.rows, out_dim.cols, out_dim.channels, out_dim.batches, "delta:");
	//Util::printDeviceData(d_input, in_dim.rows, in_dim.cols, in_dim.channels, in_dim.batches, "input:");
	_preActivation->print_grad("delta:");
	this->_input->print_data("input:");
	*/




	//if(this->name == "inception_3a/convProjection") {
	//	Data<Dtype>::printConfig = 1;
	//}


	_computePreActivationGrad();
	_computeFiltersGrad();

	//if(this->name == "inception_3a/convProjection") {
	//	exit(1);
	//}


	_computeBiasesGrad();
	_computeInputGrad();


	/*
	if(_params[0]->is_nan_grad()) {
		cout << this->name << " filter is nan grad ... " << endl;
	}
	if(_params[1]->is_nan_grad()) {
		cout << this->name << " bias is nan grad ... " << endl;
	}
	*/
}


template <typename Dtype>
void ConvLayer<Dtype>::_computePreActivationGrad() {
	this->_outputData[0]->print_grad("outputGrad:");
	this->_outputData[0]->print_data("outputData:");

	const Dtype* d_outputData = this->_outputData[0]->device_data();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	const Dtype* d_preActivationData = _preActivation->device_data();
	Dtype* d_preActivationGrad = _preActivation->mutable_device_grad();

	activation_fn->backward(this->outputTensorDesc, d_outputData, d_outputGrad, d_preActivationData, d_preActivationGrad);
}



template <typename Dtype>
void ConvLayer<Dtype>::_computeFiltersGrad() {
	this->_inputData[0]->print_data("inputData:");
	this->_preActivation->print_grad("preActivationGrad:");

	// d(Cost)/d(Filters)
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	Dtype* d_filtersGrad = _params[Filter]->mutable_device_grad();

	checkCUDNN(cudnnConvolutionBackwardFilter(Cuda::cudnnHandle,
			&Cuda::alpha, this->inputTensorDesc, d_inputData, this->outputTensorDesc, d_preActivationGrad, convDesc, convBwdFilterAlgo, d_workspace, workspaceSize,
			&Cuda::beta, filterDesc, d_filtersGrad));

	this->_params[Filter]->print_grad("filtersGrad:");
}

template <typename Dtype>
void ConvLayer<Dtype>::_computeBiasesGrad() {
	// d(Cost)/d(Biases)
	const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	Dtype* d_biasGrad = _params[Bias]->mutable_device_grad();

	checkCUDNN(cudnnConvolutionBackwardBias(Cuda::cudnnHandle,
			&Cuda::alpha, this->outputTensorDesc, d_preActivationGrad,
			&Cuda::beta, biasTensorDesc, d_biasGrad));
}

template <typename Dtype>
void ConvLayer<Dtype>::_computeInputGrad() {
	// d(Cost)/d(Input)
	const Dtype* d_filtersData = _params[Filter]->device_data();
	const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	Dtype* d_inputGrad = this->_inputData[0]->mutable_device_grad();
	checkCUDNN(cudnnConvolutionBackwardData(Cuda::cudnnHandle,
			&Cuda::alpha, filterDesc, d_filtersData, this->outputTensorDesc, d_preActivationGrad, convDesc, convBwdDataAlgo, d_workspace, workspaceSize,
			&Cuda::beta, this->inputTensorDesc, d_inputGrad));
	this->_inputData[0]->print_grad("inputGrad:");
	_params[Filter]->print_data("filtersData:");

	/*
	//if(this->name == "inception_3a/conv5x5reduce") {
	if(this->name == "inception_3a/conv1x1") {
		double grad = _params[Filter]->sumsq_device_grad();
		double data = _params[Filter]->sumsq_device_data();
		//cout << "inception_3a/conv5x5reduce grad: " << grad << ", data:" << data << endl;
		cout << "inception_3a/conv1x1 grad: " << grad << ", data:" << data << endl;
	}
	*/
}




/*
template <typename Dtype>
double ConvLayer<Dtype>::testParamAbnormality() {
	const Dtype* weightGrad = _params[Filter]->host_grad();
	const size_t count = _params[Filter]->getCount();

	double mean = 0.0;
	for(uint32_t i = 0; i < count; i++) {
		mean += weightGrad[i];
	}
	mean /= count;

	double sd = 0.0;
	for(uint32_t i = 0; i < count; i++) {
		sd += (weightGrad[i]-mean)*(weightGrad[i]-mean);
	}
	sd = sqrt(sd/(count-1));


	cout << this->name << ": mean: " << mean << ", sd: " << sd << endl;

	for(uint32_t i = 0; i < count; i++) {
		if(abs(weightGrad[i]-mean) > 10000*sd) {
			return weightGrad[i];
		}
	}
	return DBL_MAX;
}
*/








template ConvLayer<float>::~ConvLayer();
template void ConvLayer<float>::initialize(filter_dim filter_d, update_param weight_update_param, update_param bias_update_param,
		param_filler<float> weight_filler, param_filler<float> bias_filler, typename Activation<float>::Type activationType);
template void ConvLayer<float>::reshape();
template void ConvLayer<float>::_clearShape();
//template void ConvLayer<float>::_save(ofstream &ofs);
//template void ConvLayer<float>::_load(ifstream &ifs, map<Layer<float>*, Layer<float>*> &layerMap);
template void ConvLayer<float>::update();
template void ConvLayer<float>::feedforward();
template void ConvLayer<float>::backpropagation();


#endif
