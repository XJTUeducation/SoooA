#include "hip/hip_runtime.h"
/*
 * FullyConnectedLayer.cpp
 *
 *  Created on: 2016. 5. 10.
 *      Author: jhkim
 */

#include "hip/hip_runtime.h"
#include <algorithm>

#include "FullyConnectedLayer.h"
#include "Util.h"
#include "Exception.h"
#include "NetworkConfig.h"
#include "SysLog.h"
#include "StdOutLog.h"

#define FULLYCONNECTEDLAYER_LOG 0

using namespace std;

#ifdef GPU_MODE
///////////////////////////////////////////////////////////////////////////////////////////
// GPU Kernels

/**
 * Fills a floating-point array with ones.
 *
 * @param vec The array to fill.
 * @param size The number of elements in the array.
 */
template <typename Dtype>
__global__ void FillValues(Dtype *vec, int size, Dtype value)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;
	vec[idx] = value;
}

template <typename Dtype>
__global__ void DoNesterov(int size, const Dtype* dx, Dtype* v_prev, Dtype* v, Dtype* x,
    const Dtype mu, const Dtype lr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    /****
     * Nesterov Alogorithm
     *
     * v_prev = v # back this up
     * v = mu * v - learning_rate * dx # velocity update stays the same
     * x += -mu * v_prev + (1 + mu) * v # position update changes form
     *
     */

    v_prev[idx] = v[idx];
    v[idx] = mu * v[idx] - lr * dx[idx];
    x[idx] += (-1.0) * mu * v_prev[idx] + (1 + mu) * v[idx];
}

template <typename Dtype>
__global__ void DoAdagrad(int size, const Dtype* dx, Dtype* cache, Dtype* x,
    const Dtype lr, const Dtype eps)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    /****
     * Adagrad Alogorithm
     *
     * cache += dx**2
     * x += -learning_rate * dx / (sqrt(cache) + eps)
     *
     */

    cache[idx] += dx[idx] * dx[idx];
    x[idx] += (-1.0) * lr * dx[idx] / (sqrt(cache[idx]) + eps);
}

template <typename Dtype>
__global__ void DoRMSprop(int size, const Dtype* dx, Dtype* cache, Dtype* x,
    const Dtype lr, const Dtype eps, const Dtype dr)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    /****
     * RMSprop
     *
     * cache = decay_rate * cache + (1 - decay_rate) * dx**2
     * x += - learning_rate * dx / (sqrt(cache) + eps)
     *
     */

    cache[idx] = dr * cache[idx] + (1.0 - dr) * dx[idx] * dx[idx];
    x[idx] += (-1.0) * lr * dx[idx] / (sqrt(cache[idx]) + eps);
}

template <typename Dtype>
__global__ void DoAdam(int size, const Dtype* dx, Dtype* m, Dtype* v, Dtype* x,
    const Dtype lr, const Dtype eps, const Dtype beta1, const Dtype beta2)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;

    /****
     * Adam
     *
     * m = beta1 * m + (1 - beta1) * dx
     * v = beta2 * v + (1 - beta2) * (dx**2)
     * x += -learning_rate * m / (sqrt(v) + eps)
     *
     */
    m[idx] = beta1 * m[idx] + (1.0 - beta1) * dx[idx];
    v[idx] = beta2 * v[idx] + (1.0 - beta2) * dx[idx] * dx[idx];
    x[idx] += (-1.0) * lr * m[idx] / (sqrt(v[idx]) + eps);
}

///////////////////////////////////////////////////////////////////////////////////////////
// GPU Kernels

/**
 * Fills a floating-point array with ones.
 *
 * @param vec The array to fill.
 * @param size The number of elements in the array.
 */
template <typename Dtype>
__global__ void Dropout(const int n, const Dtype* in, const Dtype* mask,
		const unsigned int threashold, const float scale, Dtype *out)
{

	CUDA_KERNEL_LOOP(index, n) {
		//out[index] = in[index] * (mask[index] > threshold) * scale;
		out[index] = in[index] * (mask[index]) * scale;
	}
}

/**
 * dst array에 src array를 더한다.
 *
 * @param dst dst array, dst + src가 저장이 될 장소
 * @param src src array
 * @param N The number of elements in the array.
 */
template <typename Dtype>
__global__ void AddData(Dtype* dst, const Dtype* src, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;

	dst[idx] = dst[idx] + src[idx];
}

template <typename Dtype>
FullyConnectedLayer<Dtype>::~FullyConnectedLayer() {

    if (this->isReceiver) {
        Donator<Dtype>::releaseReceiver(this->donatorID);
    } else {
        //delete _params[ParamType::Weight];
        //delete _params[ParamType::Bias];
        //_params.clear();
        Util::clearVector(_params);

        //delete _paramsHistory[ParamType::Weight];
        //delete _paramsHistory[ParamType::Bias];
        //_paramsHistory.clear();
        Util::clearVector(_paramsHistory);
        Util::clearVector(_paramsHistory2);
    }

	//delete _preActivation;
	checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensorDesc));
	checkCudaErrors(hipFree(d_onevec));

	//ActivationFactory<Dtype>::destory(activation_fn);
}


template <typename Dtype>
void FullyConnectedLayer<Dtype>::reshape() {
	if (!Layer<Dtype>::_adjustInputShape()) {
		const uint32_t count = Util::vecCountByAxis(this->_inputShape[0], 1);
		const uint32_t inputDataCount = this->_inputData[0]->getCountByAxis(1);
		assert(count == inputDataCount);
	}



	/*
	// 배치수가 변경되는 경우는 허용하도록 하자.
	const uint32_t count = Util::vecCountByAxis(this->_inputShape[0], 1);
	const uint32_t inputDataCount = this->_inputData[0]->getCountByAxis(1);
	if (inputDataCount == count)
		return;
		*/

	// XXX: 주의


    // 여기에서는 batch 개수만 변경이 될 수 있다고 가정하였다.
    // 따라서 batch 개수에 대한 변경만 체크한다.
	if (!Layer<Dtype>::_isInputShapeChanged(0))
		return;

	const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
	uint32_t batches = inputShape[0];
	uint32_t channels = 1;
	uint32_t in_rows = this->_inputData[0]->getCountByAxis(1);
	uint32_t out_rows = this->n_out;
	uint32_t cols = 1;

	this->_inputShape[0] = {batches, channels, in_rows, cols};
	//this->_preActivation->reshape({batches, channels, out_rows, cols});
	this->_outputData[0]->reshape({batches, channels, out_rows, cols});

	checkCUDNN(hipdnnSetTensor4dDescriptor(
			this->inputTensorDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batches, channels, in_rows, cols));

	checkCUDNN(hipdnnSetTensor4dDescriptor(
			this->outputTensorDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batches, channels, out_rows, cols));

	STDOUT_COND_LOG(FULLYCONNECTEDLAYER_LOG, 
        "<%s> layer' input-0 has reshaped as: %dx%dx%dx%d\n",
        this->name.c_str(), batches, channels, in_rows, cols);
	STDOUT_COND_LOG(FULLYCONNECTEDLAYER_LOG,
	    "<%s> layer' output-0 has reshaped as: %dx%dx%dx%d\n", 
        this->name.c_str(), batches, channels, out_rows, cols);

	const uint32_t u_in = in_rows;
	const uint32_t u_out = out_rows;
	const uint32_t b_in = batches * in_rows;
	const uint32_t b_out = batches * out_rows;

	_params[ParamType::Weight]->reshape({1, 1, u_out, u_in});
	_params[ParamType::Bias]->reshape({1, u_out, 1, 1});
	_paramsHistory[ParamType::Weight]->reshape({1, 1, u_out, u_in});
	_paramsHistory[ParamType::Bias]->reshape({1, u_out, 1, 1});
	_paramsHistory2[ParamType::Weight]->reshape({1, 1, u_out, u_in});
	_paramsHistory2[ParamType::Bias]->reshape({1, u_out, 1, 1});

	if (!this->_paramsInitialized[Weight]) {
		this->weight_filler.fill(this->_params[ParamType::Weight]);
		this->_paramsInitialized[Weight] = true;
	}
	if (!this->_paramsInitialized[Bias]) {
		this->bias_filler.fill(this->_params[ParamType::Bias]);
		this->_paramsInitialized[Bias] = true;
	}

	checkCudaErrors(Util::ucudaMalloc(&this->d_onevec, sizeof(Dtype)*batches));
	//FillValues<<<RoundUp(batches, BW), BW>>>(this->d_onevec, batches, 1.0f);
	FillValues<<<SOOOA_GET_BLOCKS(batches), SOOOA_CUDA_NUM_THREADS>>>(
			this->d_onevec, batches, 1.0f);

	_mask.reshape(b_out);
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::update() {
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	const uint32_t weightSize = in_rows * out_rows;
	const Dtype regScale = this->networkConfig->_weightDecay * weight_update_param.decay_mult;
	const Dtype learnScale = 
        this->networkConfig->getLearningRate() * weight_update_param.lr_mult;
    const Dtype epsilon = this->networkConfig->_epsilon;
    const Dtype decayRate = this->networkConfig->_decayRate;
    const Dtype beta1 = this->networkConfig->_beta1;
    const Dtype beta2 = this->networkConfig->_beta2;

	_updateParam(weightSize, regScale, learnScale, epsilon, decayRate, beta1, beta2, 
        _paramsHistory[Weight], _paramsHistory2[Weight], _params[Weight]);

	const uint32_t biasSize = out_rows;
	const Dtype regScale_b = this->networkConfig->_weightDecay * bias_update_param.decay_mult;
	const Dtype learnScale_b = 
        this->networkConfig->getLearningRate() * bias_update_param.lr_mult;
	_updateParam(biasSize, regScale_b, learnScale_b, epsilon, decayRate, beta1, beta2, 
        _paramsHistory[Bias], _paramsHistory2[Bias], _params[Bias]);
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_updateParam(const uint32_t paramSize, const Dtype regScale,
    const Dtype learnScale, const Dtype epsilon, const Dtype decayRate, const Dtype beta1, 
    const Dtype beta2, Data<Dtype>* dataHistory, Data<Dtype>* dataHistory2,
    Data<Dtype>* data) {

	const uint32_t batches = this->_inputShape[0][0];
	const Dtype normScale = 1.0/batches;

	const Dtype momentum = this->networkConfig->_momentum;
	const Dtype negativeOne = -1.0;
    const Dtype negativeLearnScale = (-1.0) * learnScale;

	//Data<Dtype>::printConfig = 1;
	data->print_grad("paramGrad:");
	dataHistory->print_data("paramHistoryData:");
	dataHistory2->print_data("paramHistoryData:");
	data->print_data("paramData:");

    if (!Worker<Dtype>::isSingle())
        data->mutable_host_grad();
	Dtype* d_paramGrad = data->mutable_device_grad();
	Dtype* d_paramData = data->mutable_device_data();
	Dtype* d_paramHistoryData = dataHistory->mutable_device_data();
	Dtype* d_paramHistoryData2 = dataHistory2->mutable_device_data();

    // FIXME: ConvLayer에 동일한 코드가 있음. 추후에 정리 필요
    // (1) do normalization & regularization
    //  FIXME: 이것도 옵션으로 정규화를 할지 여부를 설정할 수 있었으면 좋겠음.
    checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize),
        &regScale, d_paramData, 1, d_paramGrad, 1));	// regularize

    // (2) apply optimizer
    Optimizer opt = this->networkConfig->_optimizer;
    if (opt == Optimizer::Momentum) {
        /****
         * Momentum Alogorithm
         *
         * v = mu * v - learning_rate * dx
         * x += v
         *
         */
    	checkCudaErrors(hipblasSscal(Cuda::cublasHandle, static_cast<int>(paramSize),
            &momentum, d_paramHistoryData, 1));				//
    	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize),
            &learnScale, d_paramGrad, 1, d_paramHistoryData, 1));		// momentum
    	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize),
            &negativeOne, d_paramHistoryData, 1, d_paramData, 1));		// update
    } else if (opt == Optimizer::Vanilla) {
        /****
         * Vanilla Alogorithm
         *
         * x += -learning_rate * dx
         *
         */
    	checkCudaErrors(hipblasSscal(Cuda::cublasHandle, static_cast<int>(paramSize),
            &learnScale, d_paramGrad, 1));				//
    	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize),
            &negativeOne, d_paramGrad, 1, d_paramData, 1));		// update
    } else if (opt == Optimizer::Nesterov) {
        /****
         * Nesterov Alogorithm
         *
         * v_prev = v # back this up
         * v = mu * v - learning_rate * dx # velocity update stays the same
         * x += -mu * v_prev + (1 + mu) * v # position update changes form
         *
         */
#if 0   // XXX: 한번 커밋후에 지울 예정..
    	checkCudaErrors(hipblasScopy(Cuda::cublasHandle, static_cast<int>(paramSize),
            &d_paramHistoryData, d_paramTempData, 1));	// v_prev = v

    	checkCudaErrors(hipblasSscal(Cuda::cublasHandle, static_cast<int>(paramSize),
            &momentum, d_paramHistoryData, 1)); // mu = mu * v
    	checkCudaErrors(hipblasSscal(Cuda::cublasHandle, static_cast<int>(paramSize),
            &learnScale, d_paramGrad, 1));      // dx = learning_rate * dx
    	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize),
            &negativeOne, d_paramGrad, 1, d_paramHistoryData, 1));		// v = -1.0 * dx + v

        const Dtype momentumPlusOne = momentum + 1.0;
    	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize),
            &momentumPlusOne, d_paramHistoryData, 1, d_paramGrad, 1));  // x += (1 + mu) * v

        const Dtype negativeMomentum = momentum * (-1.0);
    	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize),
            &negativeMomentum, d_paramTempData, 1, d_paramGrad, 1));  // x += -mu * v_prev
#else
	    DoNesterov<<<SOOOA_GET_BLOCKS(static_cast<int>(paramSize)), SOOOA_CUDA_NUM_THREADS>>>(
            static_cast<int>(paramSize), d_paramGrad, d_paramHistoryData,
            d_paramHistoryData2, d_paramData, momentum, learnScale);
#endif
    } else if (opt == Optimizer::Adagrad) {
        /****
         * Adagrad Alogorithm
         *
         * cache += dx**2
         * x += -learning_rate * dx / (sqrt(cache) + eps)
         *
         */
	    DoAdagrad<<<SOOOA_GET_BLOCKS(static_cast<int>(paramSize)), SOOOA_CUDA_NUM_THREADS>>>(
            static_cast<int>(paramSize), d_paramGrad, d_paramHistoryData,
            d_paramData, learnScale, epsilon);

    } else if (opt == Optimizer::RMSprop) {
        /****
         * RMSprop
         *
         * cache = decay_rate * cache + (1 - decay_rate) * dx**2
         * x += - learning_rate * dx / (sqrt(cache) + eps)
         *
         */
	    DoRMSprop<<<SOOOA_GET_BLOCKS(static_cast<int>(paramSize)), SOOOA_CUDA_NUM_THREADS>>>(
            static_cast<int>(paramSize), d_paramGrad, d_paramHistoryData,
            d_paramData, learnScale, epsilon, decayRate);

    } else if (opt == Optimizer::Adam) {
        /****
         * Adam
         *
         * m = beta1 * m + (1 - beta1) * dx
         * v = beta2 * v + (1 - beta2) * (dx**2)
         * x += -learning_rate * m / (sqrt(v) + eps)
         *
         */
	    DoAdam<<<SOOOA_GET_BLOCKS(static_cast<int>(paramSize)), SOOOA_CUDA_NUM_THREADS>>>(
            static_cast<int>(paramSize), d_paramGrad, d_paramHistoryData, d_paramHistoryData2,
            d_paramData, learnScale, epsilon, beta1, beta2);
    } else {
        SASSERT(false, "invalid optimizer. optimizer=%d", (int)opt);
    }

	data->print_grad("paramGrad:");
	dataHistory->print_data("paramHistoryData:");
	dataHistory2->print_data("paramHistoryData2:");
	data->print_data("paramData:");
	//Data<Dtype>::printConfig = 0;
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::applyChanges(LearnableLayer<Dtype> *targetLayer) {
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

    const uint32_t weightSize = in_rows * out_rows;
    const uint32_t biasSize = out_rows;
    FullyConnectedLayer<Dtype>* _targetLayer = (FullyConnectedLayer<Dtype>*)targetLayer;

    //int blockSize = BW;
    int blockSize = SOOOA_CUDA_NUM_THREADS;
    int gridSize;

    gridSize = (weightSize + blockSize -1) / blockSize;

    AddData<<<gridSize, blockSize>>>(
        _targetLayer->_params[Weight]->mutable_device_grad(),
        _params[Weight]->device_grad(), weightSize);

    gridSize = (biasSize + blockSize -1) / blockSize;

    AddData<<<gridSize, blockSize>>>(
        _targetLayer->_params[Bias]->mutable_device_grad(),
        _params[Bias]->device_grad(), biasSize);
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::syncParams(LearnableLayer<Dtype> *targetLayer) {
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

    const uint32_t weightSize = in_rows * out_rows;
    const uint32_t biasSize = out_rows;
    FullyConnectedLayer<Dtype>* _targetLayer = (FullyConnectedLayer<Dtype>*)targetLayer;

    memcpy(_params[Weight]->mutable_host_grad(), _targetLayer->_params[Weight]->host_grad(),
        weightSize);
    memcpy(_params[Bias]->mutable_host_grad(), _targetLayer->_params[Bias]->host_grad(),
        biasSize);
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::feedforward() {
	reshape();

	/*
	if (this->name == "fc6") {
		Data<Dtype>::printConfig = true;
		this->_inputData[0]->print_data({}, false);
		Data<Dtype>::printConfig = false;

		//exit(1);
	}
	*/

	_computeWeightedData();
	_computeWeightBiasedData();
	//_computeActivatedData();
	//_dropoutForward();

	/*
	if (this->name == "fc6") {
		Data<Dtype>::printConfig = true;
		this->_params[0]->print_data({}, false);
		this->_params[1]->print_data({}, false);
		this->_outputData[0]->print_data({}, false);
		Data<Dtype>::printConfig = false;

		exit(1);
	}
	*/


}


template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeWeightedData() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// Apply weight to input data
	const Dtype* d_weightData = _params[Weight]->device_data();
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	//Dtype* d_preActivationData = _preActivation->mutable_device_data();
	Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

	_params[Weight]->print_data();
	this->_inputData[0]->print_data();
	this->_inputData[0]->print_data_flatten();

    /**
     * [hipblasSgemm() 함수 설명 (from cuBlas User Documentation)]
     *
     * hipblasStatus_t hipblasSgemm(hipblasHandle_t handle, hipblasOperation_t transa,
     *                            hipblasOperation_t transb, int m, int n, int k, 
     *                            const float *alpha, const float *A, int * lda, 
     *                            const float *B, int ldb, const float *beta, float *C, 
     *                            int ldc)
     *
     * C = α op ( A ) op ( B ) + β C
     *
     * where α and β are scalars, and A , B and C are matrices stored in column-major format
     * with dimensions op ( A ) m × k , op ( B ) k × n and C m × n , respectively. Also, for
     * matrix A 
     *
     * op ( A ) = A if  transa == HIPBLAS_OP_N A T if  transa == HIPBLAS_OP_T A H if  transa ==
     * HIPBLAS_OP_C
     *
     * and op ( B ) is defined similarly for matrix B .
     *
     * hipblasOperation_t option
     *  (1) HIPBLAS_OP_N => the non-transpose operation is selected.
     *  (2) HIPBLAS_OP_T => the transpose operation is selected.
     *  (3) HIPBLAS_OP_C => the conjugate transpose operation is selected.
     *
     * lda,ldb,ldc => leading dimension of two-dimensional array used to store the matrix A,
     *                B, C
     */
	checkCudaErrors(hipblasSgemm(Cuda::cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			out_rows, batches, in_rows,
			&Cuda::alpha, d_weightData, out_rows, d_inputData, in_rows,
			&Cuda::beta, d_outputData, out_rows));

	//_preActivation->print_data();
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeWeightBiasedData() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// Add bias to weighted input data
	const Dtype* d_biasData = _params[Bias]->device_data();
	//Dtype* d_preActivationData = _preActivation->mutable_device_data();
	Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

	_params[Bias]->print_data();

	checkCudaErrors(hipblasSgemm(Cuda::cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			out_rows, batches, 1,
			&Cuda::alpha,
			d_biasData, out_rows,
			d_onevec, 1,
			&Cuda::alpha,
			d_outputData, out_rows));

	_params[Bias]->print_data();
}

/*
template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeActivatedData() {
	// Activate weighted sum (+ bias)
	if (activation_fn) {
		const Dtype* d_preActivationData = _preActivation->device_data();
		Dtype* d_outputData = this->_outputData[0]->mutable_device_data();
		activation_fn->forward(this->outputTensorDesc, d_preActivationData, d_outputData);
	} else {
		this->_outputData[0]->set_device_data(_preActivation);
	}

	//Data<Dtype>::printConfig = true;
	_preActivation->print_data();
	this->_outputData[0]->print_data();
	//Data<Dtype>::printConfig = false;
}
*/

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_dropoutForward() {
	// TODO skip when test
	if(this->networkConfig->_status == NetworkStatus::Train && p_dropout < 1.0f) {
		//int b_out = this->out_dim.batchsize();
		int b_out = this->_outputData[0]->getCount();
		Dtype* h_mask_mem = _mask.mutable_host_mem();

		for(int i = 0; i < b_out; i++) {
			h_mask_mem[i] = ((rand()/(RAND_MAX+1.0) > p_dropout)?1:0);
		}

		const Dtype* d_mask_mem = _mask.device_mem();
		Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

		Dropout<<<SOOOA_GET_BLOCKS(b_out), SOOOA_CUDA_NUM_THREADS>>>(
				b_out, d_outputData, d_mask_mem, 0, scale, d_outputData);
	}
}








template <typename Dtype>
void FullyConnectedLayer<Dtype>::backpropagation() {
	//_dropoutBackward();

    /*
     * 아래와 같은 simple한 network layer가 있다고 가정하자.
     *
     *               <<<< ith layer >>>>            <<<< i+1th layer >>>>
     *   .....    Xi    Wi     Ai     Fi       Yi (=Xi+1)   ........
     *                  Bi
     *   .....    O ---------  O  ------------  O            ........
     *                                                     dL/dYi is already computed
     *
     *  (※  Xi = i번째 layer의 input 값, Wi = i번째 layer의 weight, 
     *      Bi = i번째 layer의 bias 값,  Ai = i번째 layer의 중간 값
     *      Fi = i번째 layer의 activation function
     *      Yi = i번째 layer의 ouput 값, i+1 번째 layer의 input 값이기도 함
     *      L = loss, dL/dYi = i+1번째 layer에서 계산되었던 gradient 값)
     *
     *  gradient descent 방식으로 학습을 하기 위해서는 dL/dWi & dL/dBi가 필요하다.
     *  체인 룰에 의하여 아래와 같은 식으로 표현이 된다:
     *  (가) dYi/dWi = dL/dYi * dYi/dAi * dAi/dWi
     *  (나) dYi/dBi = dL/dYi * dYi/dAi * dAi/dBi
     *
     *  (가),(나)를 계산하기 위해서는 아래와 같이 4가지 계산이 필요하다.
     *
     *  (A) dL/dYi : i+1번째 layer의 backward 과정에서 _outputData[0]의 grad에 값을 저장해
     *                두었다.
     *
     *  (B) dYi/dAi : _computePreActivationGrad() 에서 dL/dYi * dYi/dAi의 계산을  수행 한다. 
     *                dL/dYi는 구해져 있기 때문에 Yi, Ai 값이 필요하다. 이 값들은 forward시에
     *                각각 _outputData[0]의 data와 _preActivation의 data에 저장이 되어 있다.
     *                activation function에 맞게 Yi, Ai, dL/dYi를 입력값으로 하여 dL/dYi * 
     *                dYi/dAi 값이 계산이 되고, 결과값은 this->_preActivation의 grad에 담는다.
     *
     *  (C) dAi/dWi : _computeWeightGrad()에서 (A), (B)의 결과를 조합하여 weight Grad를
     *               계산한다. dAi/dWi는 실제로 transpose Xi이기 때문에 GEMM 연산만 진행
     *               한다. 결과값은 _params[Weight]의 grad에 저장된다.
     *
     *  (D) dAi/dBi : (C)과정과 동일하다. _computeBiasGrad()에서 bias를 계산하고, 그 결과 값을
     *                _params[Bias]의 grad에 저장을 하는 것만 다르다.
     *
     *  마지막으로 i-1 layer에게 dL/dYi-1값을 전달해야 한다. 이 과정은 _computeInputGrad()
     *  에서 수행이 된다. 결과값을 _inputData의 grad에 저장한다. dL/dYi-1 = dL/dXi =
     *   dL/dAi * dAi/dXi가 된다. dL/dAi는 _preAcitvation의 grad에 저장이 되어 있고, dAi/dXi는
     *  Wi의 transpose 이기 때문에 계산가능하다.
     */
	_computeWeightGrad();
	_computeBiasGrad();
	_computeInputGrad();
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_dropoutBackward() {
	if(this->networkConfig->_status == NetworkStatus::Train && p_dropout < 1.0f) {
		const uint32_t batchSize = this->_inputData[0]->getCount();

		this->_outputData[0]->print_grad("outputGrad:");
		const Dtype* d_mask_mem = _mask.device_mem();
		Dtype* d_outputGrad = this->_outputData[0]->mutable_device_grad();

		Dropout<<<SOOOA_GET_BLOCKS(batchSize), SOOOA_CUDA_NUM_THREADS>>>(
				batchSize, d_outputGrad, d_mask_mem, 0, scale, d_outputGrad);

		//_mask.print("mask:");
		this->_outputData[0]->print_grad("outputGrad:");
	}
}

/*
template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computePreActivationGrad() {
	if (activation_fn) {
		const Dtype* d_y = this->_outputData[0]->device_data();
		const Dtype* d_dy = this->_outputData[0]->device_grad();
		const Dtype* d_x = this->_preActivation->device_data();
		Dtype* d_dx = this->_preActivation->mutable_device_grad();
		this->activation_fn->backward(this->outputTensorDesc, d_y, d_dy, d_x, d_dx);
	}
	else {
		this->_preActivation->set_device_grad(this->_outputData[0]);
	}

	//Data<Dtype>::printConfig = true;
	this->_outputData[0]->print_grad();
	this->_preActivation->print_grad();
	//Data<Dtype>::printConfig = false;

    //if(this->name == "softmaxLayer") {
        //double sumsq = this->_preActivation->sumsq_device_grad();
        //cout << "preActivation grad sumsq: " << sumsq << endl;
    //  Data<Dtype>::printConfig = 1;
    //  this->_preActivation->print_grad("preActivationGrad:");
    //  Data<Dtype>::printConfig = 0;
    //}
}
*/

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeWeightGrad() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// d(Cost)/d(Weight)
	//const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	Dtype* d_weightGrad = this->_params[Weight]->mutable_device_grad();

	checkCudaErrors(hipblasSgemm(Cuda::cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
			out_rows, in_rows, batches,
			&Cuda::alpha, d_outputGrad, out_rows, d_inputData, in_rows,
			&Cuda::beta, d_weightGrad, out_rows));

}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeBiasGrad() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// d(Cost)/d(Bias) (same as d_preActivationGrad)
	//const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	Dtype* d_biasGrad = _params[Bias]->mutable_device_grad();

	checkCudaErrors(hipblasSgemv(Cuda::cublasHandle, HIPBLAS_OP_N,
			out_rows, batches,
			&Cuda::alpha, d_outputGrad, out_rows, d_onevec, 1,
			&Cuda::beta, d_biasGrad, 1));
	_params[Bias]->print_grad("biasGrad:");
	_params[Weight]->print_data("weightData:");
	//_preActivation->print_grad("preActivationGrad");
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeInputGrad() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// d(Cost)/d(Input)
	const Dtype* d_weightData = _params[Weight]->device_data();
	//const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	Dtype* d_inputGrad = this->_inputData[0]->mutable_device_grad();

	checkCudaErrors(hipblasSgemm(Cuda::cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			in_rows, batches, out_rows,
			&Cuda::alpha, d_weightData, out_rows, d_outputGrad, out_rows,
			&Cuda::beta, d_inputGrad, in_rows));
	this->_inputData[0]->print_grad("inputGrad:");

	/*
	if(this->_input->is_nan_grad()) {
		cout << this->name << " _input gradient nan ... " << endl;
		Data<Dtype>::printConfig = 1;
		this->_input->print_grad("deltaInput:");
		Data<Dtype>::printConfig = 0;
		exit(1);
	}
	*/
}

template FullyConnectedLayer<float>::~FullyConnectedLayer();
template void FullyConnectedLayer<float>::reshape();
template void FullyConnectedLayer<float>::update();
template void FullyConnectedLayer<float>::feedforward();
template void FullyConnectedLayer<float>::backpropagation();

#endif
