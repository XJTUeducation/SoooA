#include "hip/hip_runtime.h"
/*
 * FullyConnectedLayer.cpp
 *
 *  Created on: 2016. 5. 10.
 *      Author: jhkim
 */

#include "FullyConnectedLayer.h"
#include "Util.h"
#include "Exception.h"
#include "NetworkConfig.h"
#include "hip/hip_runtime.h"
#include <algorithm>

#define FULLYCONNECTEDLAYER_LOG 0

using namespace std;

#ifdef GPU_MODE
///////////////////////////////////////////////////////////////////////////////////////////
// GPU Kernels

/**
 * Fills a floating-point array with ones.
 *
 * @param vec The array to fill.
 * @param size The number of elements in the array.
 */
template <typename Dtype>
__global__ void FillValues(Dtype *vec, int size, Dtype value)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= size)
		return;
	vec[idx] = value;
}


///////////////////////////////////////////////////////////////////////////////////////////
// GPU Kernels

/**
 * Fills a floating-point array with ones.
 *
 * @param vec The array to fill.
 * @param size The number of elements in the array.
 */
template <typename Dtype>
__global__ void Dropout(const int n, const Dtype* in, const Dtype* mask,
		const unsigned int threashold, const float scale, Dtype *out)
{

	CUDA_KERNEL_LOOP(index, n) {
		//out[index] = in[index] * (mask[index] > threshold) * scale;
		out[index] = in[index] * (mask[index]) * scale;
	}
}

/**
 * dst array에 src array를 더한다.
 *
 * @param dst dst array, dst + src가 저장이 될 장소
 * @param src src array
 * @param N The number of elements in the array.
 */
template <typename Dtype>
__global__ void AddArrayOfFCLayer(Dtype* dst, const Dtype* src, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= N)
		return;

	dst[idx] = dst[idx] + src[idx];
}

template <typename Dtype>
FullyConnectedLayer<Dtype>::~FullyConnectedLayer() {
	//delete _params[ParamType::Weight];
	//delete _params[ParamType::Bias];
	//_params.clear();
	Util::clearVector(_params);

	//delete _paramsHistory[ParamType::Weight];
	//delete _paramsHistory[ParamType::Bias];
	//_paramsHistory.clear();
	Util::clearVector(_paramsHistory);

	//delete _preActivation;
	checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensorDesc));
	checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensorDesc));
	checkCudaErrors(hipFree(d_onevec));

	//ActivationFactory<Dtype>::destory(activation_fn);
}


template <typename Dtype>
void FullyConnectedLayer<Dtype>::reshape() {
	if (!Layer<Dtype>::_adjustInputShape()) {
		const uint32_t count = Util::vecCountByAxis(this->_inputShape[0], 1);
		const uint32_t inputDataCount = this->_inputData[0]->getCountByAxis(1);
		assert(count == inputDataCount);
	}



	/*
	// 배치수가 변경되는 경우는 허용하도록 하자.
	const uint32_t count = Util::vecCountByAxis(this->_inputShape[0], 1);
	const uint32_t inputDataCount = this->_inputData[0]->getCountByAxis(1);
	if (inputDataCount == count)
		return;
		*/

	// XXX: 주의


	if (!Layer<Dtype>::_isInputShapeChanged(0))
		return;

	const vector<uint32_t>& inputShape = this->_inputData[0]->getShape();
	uint32_t batches = inputShape[0];
	uint32_t channels = 1;
	uint32_t in_rows = this->_inputData[0]->getCountByAxis(1);
	uint32_t out_rows = this->n_out;
	uint32_t cols = 1;

	this->_inputShape[0] = {batches, channels, in_rows, cols};
	//this->_preActivation->reshape({batches, channels, out_rows, cols});
	this->_outputData[0]->reshape({batches, channels, out_rows, cols});

	checkCUDNN(hipdnnSetTensor4dDescriptor(
			this->inputTensorDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batches, channels, in_rows, cols));

	checkCUDNN(hipdnnSetTensor4dDescriptor(
			this->outputTensorDesc,
			HIPDNN_TENSOR_NCHW,
			HIPDNN_DATA_FLOAT,
			batches, channels, out_rows, cols));

#if !FULLYCONNECTEDLAYER_LOG
	printf("<%s> layer' input-0 has reshaped as: %dx%dx%dx%d\n",
			this->name.c_str(), batches, channels, in_rows, cols);
	printf("<%s> layer' output-0 has reshaped as: %dx%dx%dx%d\n",
			this->name.c_str(), batches, channels, out_rows, cols);
#endif

	const uint32_t u_in = in_rows;
	const uint32_t u_out = out_rows;
	const uint32_t b_in = batches * in_rows;
	const uint32_t b_out = batches * out_rows;

	_params[ParamType::Weight]->reshape({1, 1, u_out, u_in});
	_params[ParamType::Bias]->reshape({1, u_out, 1, 1});
	_paramsHistory[ParamType::Weight]->reshape({1, 1, u_out, u_in});
	_paramsHistory[ParamType::Bias]->reshape({1, u_out, 1, 1});

	if (!this->_paramsInitialized[Weight]) {
		this->weight_filler.fill(this->_params[ParamType::Weight]);
		this->_paramsInitialized[Weight] = true;
	}
	if (!this->_paramsInitialized[Bias]) {
		this->bias_filler.fill(this->_params[ParamType::Bias]);
		this->_paramsInitialized[Bias] = true;
	}

	checkCudaErrors(Util::ucudaMalloc(&this->d_onevec, sizeof(Dtype)*batches));
	//FillValues<<<RoundUp(batches, BW), BW>>>(this->d_onevec, batches, 1.0f);
	FillValues<<<SOOOA_GET_BLOCKS(batches), SOOOA_CUDA_NUM_THREADS>>>(
			this->d_onevec, batches, 1.0f);

	_mask.reshape(b_out);

}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::update() {
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	const uint32_t weightSize = in_rows * out_rows;
	const Dtype regScale = this->networkConfig->_weightDecay * weight_update_param.decay_mult;
	const Dtype learnScale = 
        this->networkConfig->getLearningRate() * weight_update_param.lr_mult;
	_updateParam(weightSize, regScale, learnScale, _paramsHistory[Weight], _params[Weight]);

	const uint32_t biasSize = out_rows;
	const Dtype regScale_b = this->networkConfig->_weightDecay * bias_update_param.decay_mult;
	const Dtype learnScale_b = 
        this->networkConfig->getLearningRate() * bias_update_param.lr_mult;
	_updateParam(biasSize, regScale_b, learnScale_b, _paramsHistory[Bias], _params[Bias]);
}



template <typename Dtype>
void FullyConnectedLayer<Dtype>::_updateParam(const uint32_t paramSize, const Dtype regScale,
	const Dtype learnScale, Data<Dtype>* dataHistory, Data<Dtype>* data) {

	const uint32_t batches = this->_inputShape[0][0];
	const Dtype normScale = 1.0/batches;

	const Dtype momentum = this->networkConfig->_momentum;
	const Dtype negativeOne = -1.0;

	//Data<Dtype>::printConfig = 1;
	data->print_grad("paramGrad:");
	dataHistory->print_data("paramHistoryData:");
	data->print_data("paramData:");

    data->mutable_host_grad();
	Dtype* d_paramGrad = data->mutable_device_grad();
	Dtype* d_paramData = data->mutable_device_data();
	Dtype* d_paramHistoryData = dataHistory->mutable_device_data();

	checkCudaErrors(hipblasSscal(Cuda::cublasHandle, static_cast<int>(paramSize), &normScale,
        d_paramGrad, 1));								// normalize by batch size
	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize), &regScale,
        d_paramData, 1, d_paramGrad, 1));				// regularize
	checkCudaErrors(hipblasSscal(Cuda::cublasHandle, static_cast<int>(paramSize), &momentum,
        d_paramHistoryData, 1));						//
	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize), &learnScale,
        d_paramGrad, 1, d_paramHistoryData, 1));		// momentum
	checkCudaErrors(hipblasSaxpy(Cuda::cublasHandle, static_cast<int>(paramSize), &negativeOne,
        d_paramHistoryData, 1, d_paramData, 1));		// update

	data->print_grad("paramGrad:");
	dataHistory->print_data("paramHistoryData:");
	data->print_data("paramData:");
	//Data<Dtype>::printConfig = 0;
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::applyChanges(LearnableLayer<Dtype> *targetLayer) {
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

    const uint32_t weightSize = in_rows * out_rows;
    const uint32_t biasSize = out_rows;
    FullyConnectedLayer<Dtype>* _targetLayer = (FullyConnectedLayer<Dtype>*)targetLayer;

    //int blockSize = BW;
    int blockSize = SOOOA_CUDA_NUM_THREADS;
    int gridSize;

    gridSize = (weightSize + blockSize -1) / blockSize;

    AddArrayOfFCLayer<<<gridSize, blockSize>>>(
        _targetLayer->_params[Weight]->mutable_device_grad(),
        _params[Weight]->device_grad(), weightSize);

    gridSize = (biasSize + blockSize -1) / blockSize;

    AddArrayOfFCLayer<<<gridSize, blockSize>>>(
        _targetLayer->_params[Bias]->mutable_device_grad(),
        _params[Bias]->device_grad(), biasSize);
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::syncParams(LearnableLayer<Dtype> *targetLayer) {
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

    const uint32_t weightSize = in_rows * out_rows;
    const uint32_t biasSize = out_rows;
    FullyConnectedLayer<Dtype>* _targetLayer = (FullyConnectedLayer<Dtype>*)targetLayer;

    memcpy(_params[Weight]->mutable_host_grad(), _targetLayer->_params[Weight]->host_grad(),
        weightSize);
    memcpy(_params[Bias]->mutable_host_grad(), _targetLayer->_params[Bias]->host_grad(),
        biasSize);
#if 0
    for (uint32_t paramIdx = 0; paramIdx < weightSize; paramIdx++) {
        _params[Weight]->mutable_host_grad()[paramIdx] = 
            _targetLayer->_params[Weight]->host_grad()[paramIdx];
    }
    for (uint32_t paramIdx = 0; paramIdx < biasSize; paramIdx++) {
        _params[Bias]->mutable_host_grad()[paramIdx] = 
            _targetLayer->_params[Bias]->host_grad()[paramIdx];
    }
#endif
}


template <typename Dtype>
void FullyConnectedLayer<Dtype>::syncMutableMem() {
	_params[Weight]->mutable_device_grad();
	_params[Weight]->host_grad();
	_params[Bias]->mutable_device_grad();
	_params[Bias]->host_grad();
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::feedforward() {
	reshape();

	/*
	if (this->name == "fc6") {
		Data<Dtype>::printConfig = true;
		this->_inputData[0]->print_data({}, false);
		Data<Dtype>::printConfig = false;

		//exit(1);
	}
	*/

	_computeWeightedData();
	_computeWeightBiasedData();
	//_computeActivatedData();
	//_dropoutForward();

	/*
	if (this->name == "fc6") {
		Data<Dtype>::printConfig = true;
		this->_params[0]->print_data({}, false);
		this->_params[1]->print_data({}, false);
		this->_outputData[0]->print_data({}, false);
		Data<Dtype>::printConfig = false;

		exit(1);
	}
	*/


}


template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeWeightedData() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// Apply weight to input data
	const Dtype* d_weightData = _params[Weight]->device_data();
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	//Dtype* d_preActivationData = _preActivation->mutable_device_data();
	Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

	_params[Weight]->print_data();
	this->_inputData[0]->print_data();
	this->_inputData[0]->print_data_flatten();

	checkCudaErrors(hipblasSgemm(Cuda::cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			out_rows, batches, in_rows,
			&Cuda::alpha, d_weightData, out_rows, d_inputData, in_rows,
			&Cuda::beta, d_outputData, out_rows));

	//_preActivation->print_data();
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeWeightBiasedData() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// Add bias to weighted input data
	const Dtype* d_biasData = _params[Bias]->device_data();
	//Dtype* d_preActivationData = _preActivation->mutable_device_data();
	Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

	_params[Bias]->print_data();

	checkCudaErrors(hipblasSgemm(Cuda::cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			out_rows, batches, 1,
			&Cuda::alpha,
			d_biasData, out_rows,
			d_onevec, 1,
			&Cuda::alpha,
			d_outputData, out_rows));

	_params[Bias]->print_data();
}

/*
template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeActivatedData() {
	// Activate weighted sum (+ bias)
	if (activation_fn) {
		const Dtype* d_preActivationData = _preActivation->device_data();
		Dtype* d_outputData = this->_outputData[0]->mutable_device_data();
		activation_fn->forward(this->outputTensorDesc, d_preActivationData, d_outputData);
	} else {
		this->_outputData[0]->set_device_data(_preActivation);
	}

	//Data<Dtype>::printConfig = true;
	_preActivation->print_data();
	this->_outputData[0]->print_data();
	//Data<Dtype>::printConfig = false;
}
*/

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_dropoutForward() {
	// TODO skip when test
	if(this->networkConfig->_status == NetworkStatus::Train && p_dropout < 1.0f) {
		//int b_out = this->out_dim.batchsize();
		int b_out = this->_outputData[0]->getCount();
		Dtype* h_mask_mem = _mask.mutable_host_mem();

		for(int i = 0; i < b_out; i++) {
			h_mask_mem[i] = ((rand()/(RAND_MAX+1.0) > p_dropout)?1:0);
		}

		const Dtype* d_mask_mem = _mask.device_mem();
		Dtype* d_outputData = this->_outputData[0]->mutable_device_data();

		Dropout<<<SOOOA_GET_BLOCKS(b_out), SOOOA_CUDA_NUM_THREADS>>>(
				b_out, d_outputData, d_mask_mem, 0, scale, d_outputData);
	}
}








template <typename Dtype>
void FullyConnectedLayer<Dtype>::backpropagation() {
	//_dropoutBackward();

	//_computePreActivationGrad();
	_computeWeightGrad();
	_computeBiasGrad();
	_computeInputGrad();

}



template <typename Dtype>
void FullyConnectedLayer<Dtype>::_dropoutBackward() {
	if(this->networkConfig->_status == NetworkStatus::Train && p_dropout < 1.0f) {
		const uint32_t batchSize = this->_inputData[0]->getCount();

		this->_outputData[0]->print_grad("outputGrad:");
		const Dtype* d_mask_mem = _mask.device_mem();
		Dtype* d_outputGrad = this->_outputData[0]->mutable_device_grad();

		Dropout<<<SOOOA_GET_BLOCKS(batchSize), SOOOA_CUDA_NUM_THREADS>>>(
				batchSize, d_outputGrad, d_mask_mem, 0, scale, d_outputGrad);

		//_mask.print("mask:");
		this->_outputData[0]->print_grad("outputGrad:");
	}
}

/*
template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computePreActivationGrad() {
	if (activation_fn) {
		const Dtype* d_y = this->_outputData[0]->device_data();
		const Dtype* d_dy = this->_outputData[0]->device_grad();
		const Dtype* d_x = this->_preActivation->device_data();
		Dtype* d_dx = this->_preActivation->mutable_device_grad();
		this->activation_fn->backward(this->outputTensorDesc, d_y, d_dy, d_x, d_dx);
	}
	else {
		this->_preActivation->set_device_grad(this->_outputData[0]);
	}

	//Data<Dtype>::printConfig = true;
	this->_outputData[0]->print_grad();
	this->_preActivation->print_grad();
	//Data<Dtype>::printConfig = false;

    //if(this->name == "softmaxLayer") {
        //double sumsq = this->_preActivation->sumsq_device_grad();
        //cout << "preActivation grad sumsq: " << sumsq << endl;
    //  Data<Dtype>::printConfig = 1;
    //  this->_preActivation->print_grad("preActivationGrad:");
    //  Data<Dtype>::printConfig = 0;
    //}
}
*/

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeWeightGrad() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// d(Cost)/d(Weight)
	//const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	const Dtype* d_inputData = this->_inputData[0]->device_data();
	Dtype* d_weightGrad = this->_params[Weight]->mutable_device_grad();

	checkCudaErrors(hipblasSgemm(Cuda::cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
			out_rows, in_rows, batches,
			&Cuda::alpha, d_outputGrad, out_rows, d_inputData, in_rows,
			&Cuda::beta, d_weightGrad, out_rows));

}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeBiasGrad() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// d(Cost)/d(Bias) (same as d_preActivationGrad)
	//const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	Dtype* d_biasGrad = _params[Bias]->mutable_device_grad();

	checkCudaErrors(hipblasSgemv(Cuda::cublasHandle, HIPBLAS_OP_N,
			out_rows, batches,
			&Cuda::alpha, d_outputGrad, out_rows, d_onevec, 1,
			&Cuda::beta, d_biasGrad, 1));
	_params[Bias]->print_grad("biasGrad:");
	_params[Weight]->print_data("weightData:");
	//_preActivation->print_grad("preActivationGrad");
}

template <typename Dtype>
void FullyConnectedLayer<Dtype>::_computeInputGrad() {
	const uint32_t batches = this->_inputShape[0][0];
	const uint32_t in_rows = this->_inputShape[0][2];
	const uint32_t out_rows = this->_outputData[0]->getShape(2);

	// d(Cost)/d(Input)
	const Dtype* d_weightData = _params[Weight]->device_data();
	//const Dtype* d_preActivationGrad = this->_preActivation->device_grad();
	const Dtype* d_outputGrad = this->_outputData[0]->device_grad();
	Dtype* d_inputGrad = this->_inputData[0]->mutable_device_grad();

	checkCudaErrors(hipblasSgemm(Cuda::cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
			in_rows, batches, out_rows,
			&Cuda::alpha, d_weightData, out_rows, d_outputGrad, out_rows,
			&Cuda::beta, d_inputGrad, in_rows));
	this->_inputData[0]->print_grad("inputGrad:");

	/*
	if(this->_input->is_nan_grad()) {
		cout << this->name << " _input gradient nan ... " << endl;
		Data<Dtype>::printConfig = 1;
		this->_input->print_grad("deltaInput:");
		Data<Dtype>::printConfig = 0;
		exit(1);
	}
	*/
}

template FullyConnectedLayer<float>::~FullyConnectedLayer();
template void FullyConnectedLayer<float>::reshape();
template void FullyConnectedLayer<float>::update();
template void FullyConnectedLayer<float>::feedforward();
template void FullyConnectedLayer<float>::backpropagation();

#endif
