
#include <hip/hip_runtime.h>
/*
 * SoftmaxLayer.cpp
 *
 *  Created on: 2016. 8. 1.
 *      Author: jhkim
 */


#ifdef GPU_MODE

#include "SoftmaxLayer.h"

///////////////////////////////////////////////////////////////////////////////////////////
// GPU Kernels

/**
 * Fills a floating-point array with ones.
 *
 * @param vec The array to fill.
 * @param size The number of elements in the array.
 */
__global__ void Dropout_(const int n, const DATATYPE* in, const DATATYPE* mask,
		const unsigned int threashold, const float scale, DATATYPE *out)
{

	CUDA_KERNEL_LOOP(index, n) {
		//out[index] = in[index] * (mask[index] > threshold) * scale;
		out[index] = in[index] * (mask[index]) * scale;
	}
}



void SoftmaxLayer::backpropagation(const uint32_t* target) {
	Util::printMessage("SoftmaxLayer::target()---"+string(name));

	_target.set_mem(target, SyncMemCopyType::HostToDevice);
	//_target.print("target:");

	const DATATYPE* d_z = _preActivation->device_data();
	const DATATYPE* d_output = _output->device_data();
	const uint32_t* d_target = _target.device_mem();
	//DATATYPE* d_delta = _preActivation->mutable_device_grad();
	_output->reset_device_grad();
	DATATYPE* d_delta = _output->mutable_device_grad();
	cost_fn->backward(d_z, d_output, d_target, d_delta, out_dim.rows, out_dim.batches);

	//Util::printDeviceData(d_delta, out_dim.rows, out_dim.batches, 1, 1, "d_delta:");
	_output->print_data("d_output:");
	//_target.print("d_target:");
	_output->print_grad("d_delta:");

	_backpropagation();
	propBackpropagation();


	//_output->reset_device_grad();
	//OutputLayer::backpropagation(id, getInput(), 0);


	// Accounting for batch size in SGD
	// checkCudaErrors(cublasSscal(cublasHandle, ref_fc2.outputs * m_batchSize, &scalVal, dloss_data, 1));

	/*
	if(Util::train && p_dropout < 1.0f) {
		//Util::setPrint(true);
		Util::printDeviceData(d_delta, out_dim.rows, out_dim.batches, 1, 1, "delta_input:");
		Dropout_<<<RoundUp(out_dim.batchsize(), BW), BW>>>(out_dim.batchsize(), d_delta, d_mask, 0, scale, d_delta);


		Util::printData(mask, out_dim.rows, out_dim.batches, 1, 1, this->name+string("/mask:"));
		//DATATYPE *next_delta_input = next_layer->getDeltaInput();
		Util::printDeviceData(d_delta, out_dim.rows, out_dim.batches, 1, 1, "delta_input:");
		//Util::setPrint(false);
	}
	*/

	//Util::printDeviceData(d_input, in_dim.rows, in_dim.batches, 1, 1, "d_input:");

	/*
	_input->print_data("d_input:");
	const DATATYPE* d_input = _input->device_data();
	DATATYPE* d_delta_weight = _params[Weight]->mutable_device_grad();
	checkCudaErrors(cublasSgemm(Cuda::cublasHandle, CUBLAS_OP_N, CUBLAS_OP_T, out_dim.rows, in_dim.rows, out_dim.batches,
			&Cuda::alpha, d_delta, out_dim.rows, d_input, in_dim.rows, &Cuda::beta, d_delta_weight, out_dim.rows));
	//Util::printDeviceData(d_delta_weight, out_dim.rows, in_dim.rows, 1, 1, "d_delta_weight:");
	_params[Weight]->print_grad("d_delta_weight:");

	DATATYPE* d_delta_bias = _params[Bias]->mutable_device_grad();
	checkCudaErrors(cublasSgemv(Cuda::cublasHandle, CUBLAS_OP_N, out_dim.rows, out_dim.batches,
			&Cuda::alpha, d_delta, out_dim.rows, d_onevec, 1, &Cuda::beta, d_delta_bias, 1));
	//Util::printDeviceData(d_delta_bias, out_dim.rows, 1, 1, 1, "d_delta_bias:");
	_params[Bias]->print_grad("d_delta_bias:");

	//Util::printDeviceData(d_weight, out_dim.rows, in_dim.rows, 1, 1, "d_weight:");
	//Util::printDeviceData(d_delta, out_dim.rows, out_dim.batches, 1, 1, "d_delta:");
	_params[Weight]->print_data("d_weight:");
	_preActivation->print_grad("d_delta");

	const DATATYPE* d_weight = _params[Weight]->device_data();
	DATATYPE* d_delta_input = _input->mutable_device_grad();
	checkCudaErrors(cublasSgemm(Cuda::cublasHandle, CUBLAS_OP_T, CUBLAS_OP_N, in_dim.rows, out_dim.batches, out_dim.rows,
			&Cuda::alpha, d_weight, out_dim.rows, d_delta, out_dim.rows, &Cuda::beta, d_delta_input, in_dim.rows));

	//Util::printDeviceData(d_delta_input, in_dim.rows, in_dim.batches, 1, 1, "d_delta_input:");
	_input->print_grad("d_delta_input:");

	propBackpropagation();
	*/
}


double SoftmaxLayer::cost(const uint32_t* target) {
	// 편의상 HOST에서 계산, DEVICE 코드로 변환해야 함
	_target.set_mem(target, SyncMemCopyType::HostToHost);
	return cost_fn->forward(_output->host_data(), _target.host_mem(), out_dim.rows, out_dim.batches);
}


#endif








